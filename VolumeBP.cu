#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////////////
// VolumeBP.cu
// Developed by Dongjin Kwon
///////////////////////////////////////////////////////////////////////////////////////

///////////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2011-2014 Dongjin Kwon
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
///////////////////////////////////////////////////////////////////////////////////////

//#define CU_USE_TIMER
//#define CU_USE_CUTIL

#if defined(WIN32) || defined(WIN64)
#define WIN32_LEAN_AND_MEAN
#pragma comment(lib, "cuda.lib")
#pragma comment(lib, "cudart.lib")
#ifdef CU_USE_CUTIL
#pragma comment(lib, "cutil64.lib")
#endif
#endif
#include "stdafx.h"
//
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#ifdef CU_USE_CUTIL
#include "cutil.h"
#endif
//#include "cuPrintf.cu"

#ifndef CUDA_SAFE_CALL
#  define CUDA_SAFE_CALL_NO_SYNC( call) {                                    \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    } }

#  define CUDA_SAFE_CALL( call)     CUDA_SAFE_CALL_NO_SYNC(call);
#endif


// launch console
//#pragma comment(linker, "/entry:WinMainCRTStartup /subsystem:console")


#define CU_USE_3D_BLOCK

//#define O1_USE_OFFSET


typedef float REALV;

#define MAX_K		21
#define MAX_L8_1	81

#define MIN(a,b)  (((a) < (b)) ? (a) : (b))
#define MAX(a,b)  (((a) > (b)) ? (a) : (b))
#define TRUNCATE_MIN(a,b) { if ((a) > (b)) (a) = (b); }
#define TRUNCATE_MAX(a,b) { if ((a) < (b)) (a) = (b); }
#define TRUNCATE TRUNCATE_MIN

#define INFINITE_S 1e10


///////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////
#if CUDA_VERSION <= 4000
static __device__ REALV* ddcv;
static REALV* hOffset[3];
static __device__ REALV** dOffset;
static REALV* hRangeTerm[3];
static __device__ REALV** dRangeTerm;
static REALV* hSO1[3][3];
static REALV** ddSO1[3];
static __device__ REALV*** dSO1;
static REALV* hSO2[3][3];
static REALV** ddSO2[3];
static __device__ REALV*** dSO2;
static REALV* hSO1F2Message[3][6];
static REALV** ddSO1F2Message[3];
static __device__ REALV*** dSO1F2Message;
static REALV* hSO2F3Message[3][9];
static REALV** ddSO2F3Message[3];
static __device__ REALV*** dSO2F3Message;
static REALV* hDualMessage[3];
static __device__ REALV** dDualMessage;
//
static int mesh_x, mesh_y, mesh_z, mesh_ex, mesh_ey, mesh_ez;
static int nL, K, num_d;
static REALV gamma;
static REALV alpha_O1, d_O1;
static REALV alpha_O2, d_O2;
static REALV in_scv_w_O1F2, in_scv_w_O2F2, in_scv_w_O2F3;
static __constant__ int c_mesh_x, c_mesh_y, c_mesh_z, c_mesh_ex, c_mesh_ey, c_mesh_ez;
static __constant__ int c_nL, c_K, c_num_d;
static __constant__ REALV c_gamma;
static __constant__ REALV c_alpha_O1, c_d_O1;
static __constant__ REALV c_alpha_O2, c_d_O2;
static __constant__ REALV c_in_scv_w_O1F2, c_in_scv_w_O2F2, c_in_scv_w_O2F3;
//
static REALV disp_ex[MAX_K];
static REALV disp_ey[MAX_K];
static REALV disp_ez[MAX_K];
static __constant__ REALV c_disp_ex[MAX_K];
static __constant__ REALV c_disp_ey[MAX_K];
static __constant__ REALV c_disp_ez[MAX_K];
//
static int smode;
static __constant__ int c_smode;
//
static int L, L2, L4, L5, L6, L7, L8, L4_1, L8_1;
static __constant__ int c_L, c_L2, c_L4, c_L5, c_L6, c_L7, c_L8, c_L4_1, c_L8_1;
//
static size_t dsize;
static size_t tsize;
static int mx, my, mz;
static size_t msize;		// number of partial block
static int mmode;
//
static int threadsInX, threadsInY, threadsInZ;
static int blocksInX, blocksInY, blocksInZ, blocksInZ_4;
static dim3 Dg, Dg_4, Db;
static float invBlocksInY;
static int tsize_e;
static int Ns;
//
static unsigned long long int iLowerBound;
static __device__ unsigned long long int dLowerBound;
#else
static REALV* ddcv;
static REALV* hOffset[3];
static REALV** dOffset;
static REALV* hRangeTerm[3];
static REALV** dRangeTerm;
static REALV* hSO1[3][3];
static REALV** ddSO1[3];
static REALV*** dSO1;
static REALV* hSO2[3][3];
static REALV** ddSO2[3];
static REALV*** dSO2;
static REALV* hSO1F2Message[3][6];
static REALV** ddSO1F2Message[3];
static REALV*** dSO1F2Message;
static REALV* hSO2F3Message[3][9];
static REALV** ddSO2F3Message[3];
static REALV*** dSO2F3Message;
static REALV* hDualMessage[3];
static REALV** dDualMessage;
//
static int mesh_x, mesh_y, mesh_z, mesh_ex, mesh_ey, mesh_ez;
static int nL, K, num_d;
static REALV gamma;
static REALV alpha_O1, d_O1;
static REALV alpha_O2, d_O2;
static REALV in_scv_w_O1F2, in_scv_w_O2F2, in_scv_w_O2F3;
static __constant__ int c_mesh_x, c_mesh_y, c_mesh_z, c_mesh_ex, c_mesh_ey, c_mesh_ez;
static __constant__ int c_nL, c_K, c_num_d;
static __constant__ REALV c_gamma;
static __constant__ REALV c_alpha_O1, c_d_O1;
static __constant__ REALV c_alpha_O2, c_d_O2;
static __constant__ REALV c_in_scv_w_O1F2, c_in_scv_w_O2F2, c_in_scv_w_O2F3;
//
static REALV disp_ex[MAX_K];
static REALV disp_ey[MAX_K];
static REALV disp_ez[MAX_K];
static __constant__ REALV c_disp_ex[MAX_K];
static __constant__ REALV c_disp_ey[MAX_K];
static __constant__ REALV c_disp_ez[MAX_K];
//
static int smode;
static __constant__ int c_smode;
//
static int L, L2, L4, L5, L6, L7, L8, L4_1, L8_1;
static __constant__ int c_L, c_L2, c_L4, c_L5, c_L6, c_L7, c_L8, c_L4_1, c_L8_1;
//
static size_t dsize;
static size_t tsize;
static int mx, my, mz;
static size_t msize;		// number of partial block
static int mmode;
//
static int threadsInX, threadsInY, threadsInZ;
static int blocksInX, blocksInY, blocksInZ, blocksInZ_4;
static dim3 Dg, Dg_4, Db;
static float invBlocksInY;
static int tsize_e;
static int Ns;
//
static unsigned long long int iLowerBound;
static __device__ unsigned long long int dLowerBound;
#endif
///////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////


///////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////
#if 0
extern void Trace(const char* szFormat, ...);

void cu_Trace(const char* szFormat, ...)
{
    char szTempBuf[2048];
    va_list vlMarker;

    va_start(vlMarker, szFormat);
    vsprintf(szTempBuf, szFormat, vlMarker);
    va_end(vlMarker);

#if 0
	/*
    OutputDebugString(szTempBuf);
	/*/
	{
	    char szTempBuf2[2048];
#ifndef _DEBUG
		sprintf(szTempBuf2, "[VolReg] %s", szTempBuf);
#else
		sprintf(szTempBuf2, "%s", szTempBuf);
#endif
		OutputDebugString(szTempBuf2);
	}
	//*/
#endif
#if 1
	Trace((const char*)szTempBuf);
#endif
}
#endif

void cu_VolInit(REALV* dvol, REALV**** pvol, int ox, int oy, int oz, int mx, int my, int mz, int vd_x, int vd_y, int vd_z, int vd_s)
{
	REALV* hvol;
	int size = mx * my * mz * vd_s;
	int i, j, k, l;

	hvol = (REALV*)malloc(size * sizeof(REALV));

	for (k = oz; k < min(vd_z, oz+mz); k++) {
		for (j = oy; j < min(vd_y, oy+my); j++) {
			for (i = ox; i < min(vd_x, ox+mx); i++) {
				REALV *q = hvol + (((k-oz)*my + (j-oy))*mx + (i-ox))*vd_s;
				for (l = 0; l < vd_s; l++) {
					*q++ = pvol[k][j][i][l];
				}
			}
		}
	}

	CUDA_SAFE_CALL(hipMemcpy(dvol, hvol, size * sizeof(REALV), hipMemcpyHostToDevice));

	free(hvol);
}
void cu_VolInit(short* dvol, short**** pvol, int ox, int oy, int oz, int mx, int my, int mz, int vd_x, int vd_y, int vd_z, int vd_s)
{
	short* hvol;
	int size = mx * my * mz * vd_s;
	int i, j, k, l;

	hvol = (short*)malloc(size * sizeof(short));

	for (k = oz; k < min(vd_z, oz+mz); k++) {
		for (j = oy; j < min(vd_y, oy+my); j++) {
			for (i = ox; i < min(vd_x, ox+mx); i++) {
				short *q = hvol + (((k-oz)*my + (j-oy))*mx + (i-ox))*vd_s;
				for (l = 0; l < vd_s; l++) {
					*q++ = pvol[k][j][i][l];
				}
			}
		}
	}

	CUDA_SAFE_CALL(hipMemcpy(dvol, hvol, size * sizeof(short), hipMemcpyHostToDevice));

	free(hvol);
}

void cu_VolCopy(REALV* dvol, REALV**** pvol, int ox, int oy, int oz, int mx, int my, int mz, int vd_x, int vd_y, int vd_z, int vd_s)
{
	REALV* hvol;
	int size = mx * my * mz * vd_s;
	int i, j, k, l;

	hvol = (REALV*)malloc(size * sizeof(REALV));

	CUDA_SAFE_CALL(hipMemcpy(hvol, dvol, size * sizeof(REALV), hipMemcpyDeviceToHost));

	for (k = oz; k < min(vd_z, oz+mz); k++) {
		for (j = oy; j < min(vd_y, oy+my); j++) {
			for (i = ox; i < min(vd_x, ox+mx); i++) {
				REALV *q = hvol + (((k-oz)*my + (j-oy))*mx + (i-ox))*vd_s;
				for (l = 0; l < vd_s; l++) {
					pvol[k][j][i][l] = *q++;
				}
			}
		}
	}

	free(hvol);
}
void cu_VolCopy(REALV* dvol, REALV**** pvol, int ox, int oy, int oz, int os, int mx, int my, int mz, int ms, int vd_x, int vd_y, int vd_z, int vd_s)
{
	REALV* hvol;
	int size = mx * my * mz * ms;
	int i, j, k, l;

	hvol = (REALV*)malloc(size * sizeof(REALV));

	CUDA_SAFE_CALL(hipMemcpy(hvol, dvol, size * sizeof(REALV), hipMemcpyDeviceToHost));

	for (k = oz; k < min(vd_z, oz+mz); k++) {
		for (j = oy; j < min(vd_y, oy+my); j++) {
			for (i = ox; i < min(vd_x, ox+mx); i++) {
				REALV *q = hvol + (((k-oz)*my + (j-oy))*mx + (i-ox))*ms;
				for (l = os; l < min(vd_s, os+ms); l++) {
					//if (pvol[k][j][i][l] != *q) {
					//	TRACE2("%d %d %d %d diff %f %f\n", i, j, k, l, pvol[k][j][i][l], *q);
					//}
					pvol[k][j][i][l] = *q++;
					//pvol[k][j][i][l] = 1;
				}
			}
		}
	}

	free(hvol);
}
void cu_VolCopy(short* dvol, short**** pvol, int ox, int oy, int oz, int mx, int my, int mz, int vd_x, int vd_y, int vd_z, int vd_s)
{
	short* hvol;
	int size = mx * my * mz * vd_s;
	int i, j, k, l;

	hvol = (short*)malloc(size * sizeof(short));

	CUDA_SAFE_CALL(hipMemcpy(hvol, dvol, size * sizeof(short), hipMemcpyDeviceToHost));

	for (k = oz; k < min(vd_z, oz+mz); k++) {
		for (j = oy; j < min(vd_y, oy+my); j++) {
			for (i = ox; i < min(vd_x, ox+mx); i++) {
				short *q = hvol + (((k-oz)*my + (j-oy))*mx + (i-ox))*vd_s;
				for (l = 0; l < vd_s; l++) {
					pvol[k][j][i][l] = *q++;
				}
			}
		}
	}

	free(hvol);
}
void cu_VolCopy(short* dvol, short**** pvol, int ox, int oy, int oz, int os, int mx, int my, int mz, int ms, int vd_x, int vd_y, int vd_z, int vd_s)
{
	short* hvol;
	int size = mx * my * mz * vd_s;
	int i, j, k, l;

	hvol = (short*)malloc(size * sizeof(short));

	CUDA_SAFE_CALL(hipMemcpy(hvol, dvol, size * sizeof(short), hipMemcpyDeviceToHost));

	for (k = oz; k < min(vd_z, oz+mz); k++) {
		for (j = oy; j < min(vd_y, oy+my); j++) {
			for (i = ox; i < min(vd_x, ox+mx); i++) {
				short *q = hvol + (((k-oz)*my + (j-oy))*mx + (i-ox))*ms;
				for (l = os; l < min(vd_s, os+ms); l++) {
					pvol[k][j][i][l] = *q++;
				}
			}
		}
	}

	free(hvol);
}

extern "C"
BOOL cu_BP_Check()
{
	int i;
	hipError_t cu_res;

	cu_res = hipInit(0);
	if (cu_res != hipSuccess) {
		return FALSE;
	}

	{
		int count;
		hipDeviceProp_t prop;
		hipError_t err;
		//
		err = hipGetDeviceCount(&count);
		if (err != hipSuccess) {
			return FALSE;
		}
		//
		for (i = 0; i < count; i++) {
			CUDA_SAFE_CALL(hipGetDeviceProperties(&prop, i));
			TRACE2("GPU %d: major = %d, minor = %d, totalGlobalMem = %u, multiProcessorCount = %d, kernelExecTimeoutEnabled = %d\n", 
				i, prop.major, prop.minor, prop.totalGlobalMem, prop.multiProcessorCount, prop.kernelExecTimeoutEnabled);
		}
		for (i = 0; i < count; i++) {
			CUDA_SAFE_CALL(hipGetDeviceProperties(&prop, i));
			if (prop.major >= 2 && prop.totalGlobalMem >= 2000000000) {
				CUDA_SAFE_CALL(hipSetDevice(i));
				TRACE2("CUDA is running on GPU %d\n", i);
				return TRUE;
			}
		}
		if (i == count) {
			TRACE("There's no suitable GPU in this system\n");
			return FALSE;
		}
	}

	return FALSE;
}

extern "C"
void cu_BP_Allocate(int _mesh_x, int _mesh_y, int _mesh_z, int _mesh_ex, int _mesh_ey, int _mesh_ez,
	int _nL, int _K, int _num_d, REALV _alpha_O1, REALV _d_O1, REALV _alpha_O2, REALV _d_O2, REALV _gamma, REALV* _disp_ex, REALV* _disp_ey, REALV* _disp_ez,
	REALV _in_scv_w_O1F2, REALV _in_scv_w_O2F2, REALV _in_scv_w_O2F3)
{
	int i, j;

	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////
	hipError_t cu_res;
	TRACE2("hipInit\n");
	for (i = 0; i < 100; i++) {
		cu_res = hipInit(0);
		if (cu_res == hipSuccess) {
			break;
		} else {
			Sleep(1000);
		}
	}
	TRACE2("hipInit returns %d\n", cu_res);
	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////

	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////
	{
		int count;
		hipDeviceProp_t prop;
		hipError_t err;
		//
		TRACE2("hipGetDeviceCount\n");
		//CUDA_SAFE_CALL(hipGetDeviceCount(&count));
		for (i = 0; i < 100; i++) {
			err = hipGetDeviceCount(&count);
			if (err == hipSuccess) {
				break;
			} else {
				Sleep(1000);
			}
		}
		if (err != hipSuccess) {
			TRACE("hipGetDeviceCount returns error\n");
			exit(0);
		}
		//
		for (i = 0; i < count; i++) {
			TRACE2("hipGetDeviceProperties\n");
			CUDA_SAFE_CALL(hipGetDeviceProperties(&prop, i));
			TRACE2("GPU %d: major = %d, minor = %d, totalGlobalMem = %u, multiProcessorCount = %d, kernelExecTimeoutEnabled = %d\n", 
				i, prop.major, prop.minor, prop.totalGlobalMem, prop.multiProcessorCount, prop.kernelExecTimeoutEnabled);
		}
		for (i = 0; i < count; i++) {
			TRACE2("hipGetDeviceProperties\n");
			CUDA_SAFE_CALL(hipGetDeviceProperties(&prop, i));
			if (prop.major >= 2 && prop.totalGlobalMem >= 2000000000) {
				CUDA_SAFE_CALL(hipSetDevice(i));
				TRACE2("CUDA is running on GPU %d\n", i);
				break;
			}
		}
		if (i == count) {
			TRACE("There's no suitable GPU in this system\n");
			return;
		}
	}
	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////

	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////
	TRACE2("hipDeviceReset\n");
	CUDA_SAFE_CALL(hipDeviceReset());
	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////

	if (K > MAX_K) {
		TRACE("error: K = %d is larger than MAX_K = %d\n", K, MAX_K);
		return;
	}

	mesh_x = _mesh_x;
	mesh_y = _mesh_y;
	mesh_z = _mesh_z;
	mesh_ex = _mesh_ex;
	mesh_ey = _mesh_ey;
	mesh_ez = _mesh_ez;
	nL = _nL;
	K = _K;
	num_d = _num_d;
	//
	gamma = _gamma;
	alpha_O1 = _alpha_O1;
	d_O1 = _d_O1;
	alpha_O2 = _alpha_O2;
	d_O2 = _d_O2;
	//
	in_scv_w_O1F2 = _in_scv_w_O1F2;
	in_scv_w_O2F2 = _in_scv_w_O2F2;
	in_scv_w_O2F3 = _in_scv_w_O2F3;
	//
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_mesh_x), &mesh_x, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_mesh_y), &mesh_y, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_mesh_z), &mesh_z, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_mesh_ex), &mesh_ex, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_mesh_ey), &mesh_ey, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_mesh_ez), &mesh_ez, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_nL), &nL, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_K), &K, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_num_d), &num_d, sizeof(int)));
	//
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_gamma), &gamma, sizeof(REALV)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_alpha_O1), &alpha_O1, sizeof(REALV)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_d_O1), &d_O1, sizeof(REALV)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_alpha_O2), &alpha_O2, sizeof(REALV)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_d_O2), &d_O2, sizeof(REALV)));
	//
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_in_scv_w_O1F2), &in_scv_w_O1F2, sizeof(REALV)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_in_scv_w_O2F2), &in_scv_w_O2F2, sizeof(REALV)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_in_scv_w_O2F3), &in_scv_w_O2F3, sizeof(REALV)));
	//
	for (i = 0; i < K; i++) {
		disp_ex[i] = _disp_ex[i];
		disp_ey[i] = _disp_ey[i];
		disp_ez[i] = _disp_ez[i];
	}
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_disp_ex), &disp_ex, K * sizeof(REALV)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_disp_ey), &disp_ey, K * sizeof(REALV)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_disp_ez), &disp_ez, K * sizeof(REALV)));
	//
	if (in_scv_w_O1F2 == -2) {
		smode = 0;
	}
	if (in_scv_w_O2F3 == -2) {
		smode = 1;
	}
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_smode), &smode, sizeof(int)));
	//
	L  = K / 2;
	L2 = L * 2;
	L4 = L * 4;
	L5 = L * 5;
	L6 = L * 6;
	L7 = L * 7;
	L8 = L * 8;
	L4_1 = L * 4 + 1;
	L8_1 = L * 8 + 1;
	//
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_L), &L, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_L2), &L2, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_L4), &L4, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_L5), &L5, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_L6), &L6, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_L7), &L7, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_L8), &L8, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_L4_1), &L4_1, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_L8_1), &L8_1, sizeof(int)));
	//
	{
		size_t free_mem, total_mem;
		CUDA_SAFE_CALL(hipMemGetInfo(&free_mem, &total_mem));
		dsize = free_mem;
	}
	//
	size_t msize_xy;	// size (a slice) of dcv, Offset, RangeTerm, SO1, SO2, Messages
	mx = mesh_x;
	my = mesh_y;
	if ((in_scv_w_O1F2 != -2) && (in_scv_w_O2F3 == -2)) {
		tsize = mesh_x * mesh_y * mesh_z * (num_d + 3 + 3*K + 9*L4_1 + 18*K + 3*K) * sizeof(REALV);
		//msize_xy = mesh_x * mesh_y * (num_d + 3 + 3*K + 9*L4_1 + 18*K + 3*K) * sizeof(REALV);
		msize_xy = mesh_x * mesh_y * max(num_d/4 + 3 + 3*K + 18*K + 3*K, 3 + 3*K + 3*L4_1 + 18*K + 3*K) * sizeof(REALV);
	} else if ((in_scv_w_O1F2 == -2) && (in_scv_w_O2F3 != -2)) {
		tsize = mesh_x * mesh_y * mesh_z * (num_d + 3 + 3*K + 9*L8_1 + 27*K + 3*K) * sizeof(REALV);
		//msize_xy = mesh_x * mesh_y * (num_d + 3 + 3*K + 9*L8_1 + 27*K + 3*K) * sizeof(REALV);
		msize_xy = mesh_x * mesh_y * max(num_d/4 + 3 + 3*K + 27*K + 3*K, 3 + 3*K + 3*L8_1 + 27*K + 3*K) * sizeof(REALV);
	} else if ((in_scv_w_O1F2 != -2) && (in_scv_w_O2F3 != -2)) {
		tsize = mesh_x * mesh_y * mesh_z * (num_d + 3 + 3*K + (9*L4_1 + 18*K) + (9*L8_1 + 27*K) + 3*K) * sizeof(REALV);
		//msize_xy = mesh_x * mesh_y * (num_d + 3 + 3*K + (9*L4_1 + 18*K) + (9*L8_1 + 27*K) + 3*K) * sizeof(REALV);
		msize_xy = mesh_x * mesh_y * max(num_d/4 + 3 + 3*K + 27*K + 3*K, 3 + 3*K + (3*L4_1 + 18*K) + (3*L8_1 + 27*K) + 3*K) * sizeof(REALV);
	}
	//
	tsize += 1000000000;
	//
	mz = min((int)(dsize / msize_xy), mesh_z);
	if (mz < mesh_z) {
		TRACE("mz = %d, mesh_z = %d\n", mz, mesh_z);
		return;
	}
	msize = mx * my * mz;
	if (tsize >= dsize) {
		mmode = 1;
	} else {
		mmode = 0;
	}
	TRACE2("tsize = %u, dsize = %u -> mmode = %d\n", tsize, dsize, mmode);

	/*
	TRACE2("dsize = %d\n", dsize);
	TRACE2("bsize_xy = %d\n", bsize_xy);
	TRACE2("msize_xy = %d\n", msize_xy);
	TRACE2("mz = %d\n", mz);
	TRACE2("msize = %d\n", msize);
	*/


	///////////////////////////////////////////////////////////////////////////////////////
	// Setting grid and block sizes
	///////////////////////////////////////////////////////////////////////////////////////
	threadsInX = 8;
	threadsInY = 8;
	threadsInZ = 1;
	blocksInX   = (mesh_x   + threadsInX-1) / threadsInX;
	blocksInY   = (mesh_y   + threadsInY-1) / threadsInY;
	blocksInZ   = (mz       + threadsInZ-1) / threadsInZ;
	blocksInZ_4 = ((mz / 4) + threadsInZ-1) / threadsInZ;
	Dg   = dim3(blocksInX, blocksInY, blocksInZ);
	Dg_4 = dim3(blocksInX, blocksInY, blocksInZ_4);
	Db   = dim3(threadsInX, threadsInY, threadsInZ);
	invBlocksInY = 1.0f / (float)blocksInY;
	//tsize_e = 6*K + num_d;
	tsize_e = 4*K;
	Ns = threadsInX * threadsInY * threadsInZ * tsize_e * sizeof(REALV);
	//
	//CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferNone));
	//
	if (Ns > 16000) {
		TRACE("error: Ns = %d is larger than 16000\n", Ns);
		return;
	}

	/*
	TRACE2("Db = %d, %d, %d\n", threadsInX, threadsInY, threadsInZ);
	TRACE2("Dg = %d, %d, %d\n", blocksInX, blocksInY*blocksInZ, 1);
	*/
	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////

#ifdef CU_USE_TIMER
#ifdef CU_USE_CUTIL
	unsigned int timer;
	cutCreateTimer(&timer);
	cutStartTimer(timer);
#else
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif
#endif

	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////
	if (mmode == 0) {
		CUDA_SAFE_CALL(hipMalloc(&ddcv, msize * num_d * sizeof(REALV)));
	}
	//
	CUDA_SAFE_CALL(hipMalloc(&dOffset, 3 * sizeof(REALV*)));
	CUDA_SAFE_CALL(hipMalloc(&dRangeTerm, 3 * sizeof(REALV*)));
	CUDA_SAFE_CALL(hipMalloc(&dDualMessage, 3 * sizeof(REALV*)));
	//
	CUDA_SAFE_CALL(hipMalloc(&dSO1, 3 * sizeof(REALV**)));
	CUDA_SAFE_CALL(hipMalloc(&dSO1F2Message, 3 * sizeof(REALV**)));
	//
	CUDA_SAFE_CALL(hipMalloc(&dSO2, 3 * sizeof(REALV**)));
	CUDA_SAFE_CALL(hipMalloc(&dSO2F3Message, 3 * sizeof(REALV**)));
	//
	for (i = 0; i < 3; i++) {
		CUDA_SAFE_CALL(hipMalloc(&hOffset[i], msize * sizeof(REALV)));
		CUDA_SAFE_CALL(hipMalloc(&hRangeTerm[i], msize * K * sizeof(REALV)));
		CUDA_SAFE_CALL(hipMalloc(&hDualMessage[i], msize * K * sizeof(REALV)));
		//
		if (in_scv_w_O1F2 != -2) {
			CUDA_SAFE_CALL(hipMalloc(&ddSO1[i], 3 * sizeof(REALV*)));
			if (mmode == 0) {
				for (j = 0; j < 3; j++) {
					CUDA_SAFE_CALL(hipMalloc(&hSO1[i][j], msize * L4_1 * sizeof(REALV)));
				}
				CUDA_SAFE_CALL(hipMemcpy(ddSO1[i], hSO1[i], 3 * sizeof(REALV*), hipMemcpyHostToDevice));
			}
			//
			CUDA_SAFE_CALL(hipMalloc(&ddSO1F2Message[i], 6 * sizeof(REALV*)));
			for (j = 0; j < 6; j++) {
				CUDA_SAFE_CALL(hipMalloc(&hSO1F2Message[i][j], msize * K * sizeof(REALV)));
			}
			CUDA_SAFE_CALL(hipMemcpy(ddSO1F2Message[i], hSO1F2Message[i], 6 * sizeof(REALV*), hipMemcpyHostToDevice));
		}
		if (in_scv_w_O2F3 != -2) {
			CUDA_SAFE_CALL(hipMalloc(&ddSO2[i], 3 * sizeof(REALV*)));
			if (mmode == 0) {
				for (j = 0; j < 3; j++) {
					CUDA_SAFE_CALL(hipMalloc(&hSO2[i][j], msize * L8_1 * sizeof(REALV)));
				}
				CUDA_SAFE_CALL(hipMemcpy(ddSO2[i], hSO2[i], 3 * sizeof(REALV*), hipMemcpyHostToDevice));
			}
			//
			CUDA_SAFE_CALL(hipMalloc(&ddSO2F3Message[i], 9 * sizeof(REALV*)));
			for (j = 0; j < 9; j++) {
				CUDA_SAFE_CALL(hipMalloc(&hSO2F3Message[i][j], msize * K * sizeof(REALV)));
			}
			CUDA_SAFE_CALL(hipMemcpy(ddSO2F3Message[i], hSO2F3Message[i], 9 * sizeof(REALV*), hipMemcpyHostToDevice));
		}
	}
	CUDA_SAFE_CALL(hipMemcpy(dRangeTerm, hRangeTerm, 3 * sizeof(REALV*), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dDualMessage, hDualMessage, 3 * sizeof(REALV*), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dOffset, hOffset, 3 * sizeof(REALV*), hipMemcpyHostToDevice));
	if (in_scv_w_O1F2 != -2) {
		CUDA_SAFE_CALL(hipMemcpy(dSO1F2Message, ddSO1F2Message, 3 * sizeof(REALV**), hipMemcpyHostToDevice));
	}
	if (in_scv_w_O2F3 != -2) {
		CUDA_SAFE_CALL(hipMemcpy(dSO2F3Message, ddSO2F3Message, 3 * sizeof(REALV**), hipMemcpyHostToDevice));
	}
	if (mmode == 0) {
		if (in_scv_w_O1F2 != -2) {
			CUDA_SAFE_CALL(hipMemcpy(dSO1, ddSO1, 3 * sizeof(REALV**), hipMemcpyHostToDevice));
		}
		if (in_scv_w_O2F3 != -2) {
			CUDA_SAFE_CALL(hipMemcpy(dSO2, ddSO2, 3 * sizeof(REALV**), hipMemcpyHostToDevice));
		}
	}
	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////

#ifdef CU_USE_TIMER
#ifdef CU_USE_CUTIL
	cutStopTimer(timer);
	TRACE2("alloc time = %f\n", cutGetTimerValue(timer));
	cutDeleteTimer(timer);
#else
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	TRACE2("alloc time = %f\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
#endif
#endif
}

extern "C"
void cu_BP_Free()
{
	int i, j;

	hipDeviceSynchronize();

#ifdef CU_USE_TIMER
#ifdef CU_USE_CUTIL
	unsigned int timer;
	cutCreateTimer(&timer);
	cutStartTimer(timer);
#else
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif
#endif

	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////
	if (mmode == 0) {
		CUDA_SAFE_CALL(hipFree(ddcv));
	}
	//
	for (i = 0; i < 3; i++) {
		CUDA_SAFE_CALL(hipFree(hOffset[i]));
		CUDA_SAFE_CALL(hipFree(hRangeTerm[i]));
		CUDA_SAFE_CALL(hipFree(hDualMessage[i]));
		//
		if (in_scv_w_O1F2 != -2) {
			if (mmode == 0) {
				for (j = 0; j < 3; j++) {
					CUDA_SAFE_CALL(hipFree(hSO1[i][j]));
				}
			}
			for (j = 0; j < 6; j++) {
				CUDA_SAFE_CALL(hipFree(hSO1F2Message[i][j]));
			}
			CUDA_SAFE_CALL(hipFree(ddSO1[i]));
			CUDA_SAFE_CALL(hipFree(ddSO1F2Message[i]));
		}
		if (in_scv_w_O2F3 != -2) {
			if (mmode == 0) {
				for (j = 0; j < 3; j++) {
					CUDA_SAFE_CALL(hipFree(hSO2[i][j]));
				}
			}
			for (j = 0; j < 9; j++) {
				CUDA_SAFE_CALL(hipFree(hSO2F3Message[i][j]));
			}
			CUDA_SAFE_CALL(hipFree(ddSO2[i]));
			CUDA_SAFE_CALL(hipFree(ddSO2F3Message[i]));
		}
	}
	CUDA_SAFE_CALL(hipFree(dOffset));
	CUDA_SAFE_CALL(hipFree(dRangeTerm));
	CUDA_SAFE_CALL(hipFree(dDualMessage));
	//
	CUDA_SAFE_CALL(hipFree(dSO1));
	CUDA_SAFE_CALL(hipFree(dSO1F2Message));
	//
	CUDA_SAFE_CALL(hipFree(dSO2));
	CUDA_SAFE_CALL(hipFree(dSO2F3Message));
	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////

#ifdef CU_USE_TIMER
#ifdef CU_USE_CUTIL
	cutStopTimer(timer);
	TRACE2("free time = %f\n", cutGetTimerValue(timer));
	cutDeleteTimer(timer);
#else
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	TRACE2("free time = %f\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
#endif
#endif
}
///////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////


///////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////
__device__ static void cu_Add2Message(REALV* message, const REALV* other, int nstates)
{
	int i;
	for (i = 0; i < nstates; i++) {
		message[i] += other[i];
	}
}
#if 0
__device__ static REALV cu_FindMin(REALV* message, int nstates)
{
	REALV min;
	int i;
	min = message[0];
	for (i = 1; i < nstates; i++) {
		if (min > message[i]) {
			min = message[i];
		}
	}
	return min;
}
#endif
__device__ static void cu_SubtractMin(REALV* message, int nstates, REALV& min)
{
	int i;
	min = message[0];
	for (i = 1; i < nstates; i++) {
		if (min > message[i]) {
			min = message[i];
		}
	}
	for (i = 0; i < nstates; i++) {
		message[i] -= min;
	}
}
__device__ static void cu_SubtractMin(REALV* message, REALV* message_out, int nstates, REALV& min)
{
	int i;
	min = message[0];
	for (i = 1; i < nstates; i++) {
		if (min > message[i]) {
			min = message[i];
		}
	}
	for (i = 0; i < nstates; i++) {
		message_out[i] = message[i] - min;
	}
}
__device__ static void cu_Add2MessageDual(REALV* message, int x, int y, int z, int nStates,
	size_t idx, REALV* dRangeTerm, REALV** dSO1F2Message, REALV** dSO2F3Message)
{
	// initialize the message using the range term
	memcpy(message, &dRangeTerm[idx*nStates], sizeof(REALV) * nStates);

	// add spatial messages
	if (c_in_scv_w_O1F2 != -2) {
		if (x > 0) {			// add x- -> x+
			cu_Add2Message(message, &dSO1F2Message[0][idx*nStates], nStates);
		}
		if (x < c_mesh_x-1) {	// add x+ -> x-
			cu_Add2Message(message, &dSO1F2Message[1][idx*nStates], nStates);
		}
		if (y > 0) {			// add y- -> y+
			cu_Add2Message(message, &dSO1F2Message[2][idx*nStates], nStates);
		}
		if (y < c_mesh_y-1) {	// add y+ -> y-
			cu_Add2Message(message, &dSO1F2Message[3][idx*nStates], nStates);
		}
		if (z > 0) {			// add z- -> z+
			cu_Add2Message(message, &dSO1F2Message[4][idx*nStates], nStates);
		}
		if (z < c_mesh_z-1) {	// add z+ -> z-
			cu_Add2Message(message, &dSO1F2Message[5][idx*nStates], nStates);
		}
	}
	if (c_in_scv_w_O2F3 != -2) {
		if (x < c_mesh_x-2) {			// f+ -> x
			cu_Add2Message(message, &dSO2F3Message[0][idx*nStates], nStates);
		}
		if (x > 0 && x < c_mesh_x-1) {	// f0 -> x
			cu_Add2Message(message, &dSO2F3Message[1][idx*nStates], nStates);
		}
		if (x > 1) {					// f- -> x
			cu_Add2Message(message, &dSO2F3Message[2][idx*nStates], nStates);
		}
		if (y < c_mesh_y-2) {
			cu_Add2Message(message, &dSO2F3Message[3][idx*nStates], nStates);
		}
		if (y > 0 && y < c_mesh_y-1) {
			cu_Add2Message(message, &dSO2F3Message[4][idx*nStates], nStates);
		}
		if (y > 1) {
			cu_Add2Message(message, &dSO2F3Message[5][idx*nStates], nStates);
		}
		if (z < c_mesh_z-2) {
			cu_Add2Message(message, &dSO2F3Message[6][idx*nStates], nStates);
		}
		if (z > 0 && z < c_mesh_z-1) {
			cu_Add2Message(message, &dSO2F3Message[7][idx*nStates], nStates);
		}
		if (z > 1) {
			cu_Add2Message(message, &dSO2F3Message[8][idx*nStates], nStates);
		}
	}
}
__device__ static void cu_Add2MessageSpatial_O1F2(REALV* message, int x, int y, int z, int direction, int nStates,
	size_t idx, REALV* dRangeTerm, REALV** dSO1F2Message, REALV** dSO2F3Message, REALV* dDualMessage)
{
	// initialize the message from the dual plane
	memcpy(message, &dDualMessage[idx*nStates], sizeof(REALV) * nStates);

	// add the range term
	cu_Add2Message(message, &dRangeTerm[idx*nStates], nStates);

	// add spatial messages
	if (c_in_scv_w_O1F2 != -2) {
		if (x > 0          && direction != 1) {	// add x- -> x+
			cu_Add2Message(message, &dSO1F2Message[0][idx*nStates], nStates);
		}
		if (x < c_mesh_x-1 && direction != 0) {	// add x+ -> x-
			cu_Add2Message(message, &dSO1F2Message[1][idx*nStates], nStates);
		}
		if (y > 0          && direction != 3) {	// add y- -> y+
			cu_Add2Message(message, &dSO1F2Message[2][idx*nStates], nStates);
		}
		if (y < c_mesh_y-1 && direction != 2) {	// add y+ -> y-
			cu_Add2Message(message, &dSO1F2Message[3][idx*nStates], nStates);
		}
		if (z > 0          && direction != 5) {	// add z- -> z+
			cu_Add2Message(message, &dSO1F2Message[4][idx*nStates], nStates);
		}
		if (z < c_mesh_z-1 && direction != 4) {	// add z+ -> z-
			cu_Add2Message(message, &dSO1F2Message[5][idx*nStates], nStates);
		}
	}
	if (c_in_scv_w_O2F3 != -2) {
		if (x < c_mesh_x-2) {			// f+ -> x
			cu_Add2Message(message, &dSO2F3Message[0][idx*nStates], nStates);
		}
		if (x > 0 && x < c_mesh_x-1) {	// f0 -> x
			cu_Add2Message(message, &dSO2F3Message[1][idx*nStates], nStates);
		}
		if (x > 1) {					// f- -> x
			cu_Add2Message(message, &dSO2F3Message[2][idx*nStates], nStates);
		}
		if (y < c_mesh_y-2) {
			cu_Add2Message(message, &dSO2F3Message[3][idx*nStates], nStates);
		}
		if (y > 0 && y < c_mesh_y-1) {
			cu_Add2Message(message, &dSO2F3Message[4][idx*nStates], nStates);
		}
		if (y > 1) {
			cu_Add2Message(message, &dSO2F3Message[5][idx*nStates], nStates);
		}
		if (z < c_mesh_z-2) {
			cu_Add2Message(message, &dSO2F3Message[6][idx*nStates], nStates);
		}
		if (z > 0 && z < c_mesh_z-1) {
			cu_Add2Message(message, &dSO2F3Message[7][idx*nStates], nStates);
		}
		if (z > 1 && direction != 8) {
			cu_Add2Message(message, &dSO2F3Message[8][idx*nStates], nStates);
		}
	}
}
__device__ static void cu_Add2MessageSpatial_O2F3(REALV* message, int x, int y, int z, int direction, int nStates,
	size_t idx, REALV* dRangeTerm, REALV** dSO1F2Message, REALV** dSO2F3Message, REALV* dDualMessage)
{
	// initialize the message from the dual plane
	memcpy(message, &dDualMessage[idx*nStates], sizeof(REALV) * nStates);

	// add the range term
	cu_Add2Message(message, &dRangeTerm[idx*nStates], nStates);

	// add spatial messages
	if (c_in_scv_w_O1F2 != -2) {
		if (x > 0) {			// add x- -> x+
			cu_Add2Message(message, &dSO1F2Message[0][idx*nStates], nStates);
		}
		if (x < c_mesh_x-1) {	// add x+ -> x-
			cu_Add2Message(message, &dSO1F2Message[1][idx*nStates], nStates);
		}
		if (y > 0) {			// add y- -> y+
			cu_Add2Message(message, &dSO1F2Message[2][idx*nStates], nStates);
		}
		if (y < c_mesh_y-1) {	// add y+ -> y-
			cu_Add2Message(message, &dSO1F2Message[3][idx*nStates], nStates);
		}
		if (z > 0) {			// add z- -> z+
			cu_Add2Message(message, &dSO1F2Message[4][idx*nStates], nStates);
		}
		if (z < c_mesh_z-1) {	// add z+ -> z-
			cu_Add2Message(message, &dSO1F2Message[5][idx*nStates], nStates);
		}
	}
	if (c_in_scv_w_O2F3 != -2) {
		if (x < c_mesh_x-2 && direction != 0) {				// f+ -> x
			cu_Add2Message(message, &dSO2F3Message[0][idx*nStates], nStates);
		}
		if (x > 0 && x < c_mesh_x-1 && direction != 1) {	// f0 -> x
			cu_Add2Message(message, &dSO2F3Message[1][idx*nStates], nStates);
		}
		if (x > 1 && direction != 2) {						// f- -> x
			cu_Add2Message(message, &dSO2F3Message[2][idx*nStates], nStates);
		}
		if (y < c_mesh_y-2 && direction != 3) {
			cu_Add2Message(message, &dSO2F3Message[3][idx*nStates], nStates);
		}
		if (y > 0 && y < c_mesh_y-1 && direction != 4) {
			cu_Add2Message(message, &dSO2F3Message[4][idx*nStates], nStates);
		}
		if (y > 1 && direction != 5) {
			cu_Add2Message(message, &dSO2F3Message[5][idx*nStates], nStates);
		}
		if (z < c_mesh_z-2 && direction != 6) {
			cu_Add2Message(message, &dSO2F3Message[6][idx*nStates], nStates);
		}
		if (z > 0 && z < c_mesh_z-1 && direction != 7) {
			cu_Add2Message(message, &dSO2F3Message[7][idx*nStates], nStates);
		}
		if (z > 1 && direction != 8) {
			cu_Add2Message(message, &dSO2F3Message[8][idx*nStates], nStates);
		}
	}
}
__device__ void cu_ComputeSpatialMessageDT(REALV* message, REALV* message_org, REALV* message_buf, int x, int y, int z, REALV d0, int nStates, int wsize, REALV* c_disp_e)
{
#if 0
	//////////////////////////////////////////
	REALV Min;
	ptrdiff_t l;

	if (message_org != message_buf) {
		memcpy(message_buf, message_org, nStates * sizeof(REALV));
	}

	// use distance transform function to impose smoothness compatibility
	Min = cu_FindMin(message_buf, nStates) + c_d;
	for (l = 1; l < nStates; l++) {
		message_buf[l] = min(message_buf[l], message_buf[l-1] + c_alpha);
	}
	for (l = nStates-2; l >= 0; l--) {
		message_buf[l] = min(message_buf[l], message_buf[l+1] + c_alpha);
	}

	// transform the compatibility 
	int shift = -d0;
	if (abs(shift) > wsize+wsize) { // the shift is too big that there is no overlap
		if (x > 0 || y > 0 || z > 0) {
			for (l = 0; l < nStates; l++) {
				message[l] =  l * c_alpha;
			}
		} else {
			for (l = 0; l < nStates; l++) {
				message[l] = -l * c_alpha;
			}
		}
	} else {
		int start = max(-wsize, shift-wsize);
		int end   = min( wsize, shift+wsize);
		for (l = start; l <= end; l++) {
			message[l-shift+wsize] = message_buf[l+wsize];
		}
		if (start-shift+wsize > 0) {
			for (l = start-shift+wsize-1; l >= 0; l--) {
				message[l] = message[l+1] + c_alpha;
			}
		}
		if (end-shift+wsize < nStates) {
			for (l = end-shift+wsize+1; l < nStates; l++) {
				message[l] = message[l-1] + c_alpha;
			}
		}
	}

	// put back the threshold
	for (l = 0; l < nStates; l++) {
		message[l] = min(message[l], Min);
	}
	//////////////////////////////////////////
#endif
	//
#if 1
	//////////////////////////////////////////
	REALV s, T;
	REALV v_fx, delta_1; //delta_f
	int k0, k1;
	//
	s = c_alpha_O1;
	T = c_d_O1;
	//////////////////////////////////////////
	for (k1 = 0; k1 < nStates; k1++) {
		delta_1 = min(s * fabs(d0+c_disp_e[0]-c_disp_e[k1]), T) + message_org[0];
		for (k0 = 0; k0 < nStates; k0++) {
			v_fx = min(s * fabs(d0+c_disp_e[k0]-c_disp_e[k1]), T) + message_org[k0];
			TRUNCATE(delta_1, v_fx);
		}
		message[k1] = delta_1;
	}
	//////////////////////////////////////////
#endif
	//
#if 0
	//////////////////////////////////////////
	REALV s, T;
	REALV _2s, xv;
	REALV fx0_min, fx0_min_T;
	int k, k0, k1;
	REALV zv[MAX_K+2];
	int v[MAX_K];
	REALV v_fx, delta_1;
	//
	s = c_alpha;
	T = c_d;
	_2s = 0.5f / s;
	//////////////////////////////////////////
	fx0_min = message_org[0];
	for (k0 = 1; k0 < nStates; k0++) {
		TRUNCATE(fx0_min, message_org[k0]);
	}
	fx0_min_T = fx0_min + T;
	//////////////////////////////////////////
	// DT
	k = 0;
	v[0] = 0;
	zv[0] = -INFINITE_S;
	zv[1] = INFINITE_S;
	for (k1 = 1; k1 < nStates; k1++) {
		xv = ((message_org[k1] + s*c_disp_e[k1]) - (message_org[v[k]] - s*c_disp_e[v[k]])) * _2s;
		if (xv > c_disp_e[v[k]] && xv < c_disp_e[k1]) {
			if (xv <= zv[k]) {
				k--;
				//
				k1--;
				continue;
			} else {
				k++;
				//
				v[k] = k1;
				zv[k] = xv;
				zv[k+1] = INFINITE_S;
			}
		} else if ((xv == c_disp_e[v[k]]) || (xv == c_disp_e[k1])) {
			if (message_org[k1] < message_org[v[k]]) {
				v[k] = k1;
				zv[k+1] = INFINITE_S;
			}
		} else {
			if (k == 0) {
				if (message_org[k1] < message_org[v[0]]) {
					v[0] = k1;
					zv[0] = -INFINITE_S;
					zv[1] = INFINITE_S;
				}
			} else {
				if (message_org[k1] < message_org[v[k]]) {
					k--;
					//
					k1--;
					continue;
				}
			}
		}
	}
	k = 0;
	for (k1 = 0; k1 < nStates; k1++) {
		while (zv[k+1] < -d0 + c_disp_e[k1]) {
			k++;
		}
		delta_1 = min(s * fabs(d0+c_disp_e[v[k]]-c_disp_e[k1]), T) + message_org[v[k]];
		// apply truncation
		message[k1] = min(delta_1, fx0_min_T);
	}
	//////////////////////////////////////////
#endif
}
///////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////


///////////////////////////////////////////////////////////////////////////////////////
// Update Message for BP
///////////////////////////////////////////////////////////////////////////////////////
__device__ static void cu_UpdateSpatialMessageD(int x, int y, int z, int direction,
	int cx, int cy, int cz, int ox, int oy, int oz, int mx, int my, int mz,
	REALV* dOffset, REALV* dRangeTerm, REALV** dSO1F2Message, REALV* dDualMessage)
{
	int x1, y1, z1;
	REALV* message_org;
	//REALV message_org[MAX_K];
	REALV* message;
	size_t idx, idx1;
	REALV d0, Min;
	REALV* c_disp_e;
	//
	extern __shared__ REALV sbuf[];

	// eliminate impossible messages
	if (direction == 0 && x == c_mesh_x-1) { return; }
	if (direction == 1 && x == 0         ) { return; }
	if (direction == 2 && y == c_mesh_y-1) { return; }
	if (direction == 3 && y == 0         ) { return; }
	if (direction == 4 && z == c_mesh_z-1) { return; }
	if (direction == 5 && z == 0         ) { return; }

	x1 = x; y1 = y; z1 = z; // get the destination
	switch (direction) {
	case 0: 
		x1++; 
		c_disp_e = c_disp_ex;
		break;
	case 1: 
		x1--; 
		c_disp_e = c_disp_ex;
		break;
	case 2: 
		y1++; 
		c_disp_e = c_disp_ey;
		break;
	case 3: 
		y1--; 
		c_disp_e = c_disp_ey;
		break;
	case 4: 
		z1++; 
		c_disp_e = c_disp_ez;
		break;
	case 5: 
		z1--; 
		c_disp_e = c_disp_ez;
		break;
	}

	idx  = ((cz   )*my + (cy   ))*mx + (cx   );
	idx1 = ((z1-oz)*my + (y1-oy))*mx + (x1-ox);

	message_org = &sbuf[(threadIdx.y * blockDim.x + threadIdx.x)*c_K];
	message     = &dSO1F2Message[direction][idx1*c_K];

	cu_Add2MessageSpatial_O1F2(message_org, x, y, z, direction, c_K, idx, dRangeTerm, dSO1F2Message, NULL, dDualMessage);

	__syncthreads();

#ifdef O1_USE_OFFSET
	d0 = dOffset[idx] - dOffset[idx1];
#else
	d0 = 0;
#endif

	cu_ComputeSpatialMessageDT(message, message_org, message_org, x, y, z, d0, c_K, c_nL, c_disp_e);
	// normalize the message by subtracting the minimum value
	cu_SubtractMin(message, c_K, Min);
}

__global__ static void cu_UpdateSpatialMessage(unsigned int blocksInY, float invBlocksInY, int bw, int ox, int oy, int oz, int mx, int my, int mz,
	REALV* dOffset, REALV* dRangeTerm, REALV** dSO1F2Message, REALV* dDualMessage)
{
	int cx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
#ifdef CU_USE_3D_BLOCK
	int cy = __umul24(blockIdx.y , blockDim.y) + threadIdx.y;
	int cz = __umul24(blockIdx.z , blockDim.z) + threadIdx.z;
#else
	int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
	int blockIdxy = blockIdx.y - __umul24(blockIdxz, blocksInY);
	int cy = __umul24(blockIdxy , blockDim.y) + threadIdx.y;
	int cz = __umul24(blockIdxz , blockDim.z) + threadIdx.z;
#endif
	int x, y, z;

	//cuPrintf("\tbx = %3d, by = %3d, bz = %3d, tx = %3d, ty = %3d, tz = %3d, idx = (%3d, %3d, %3d)\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z, cx, cy, cz);

	x = cx + ox;
	y = cy + oy;
	z = cz + oz;

	if (bw == 0) {
		cu_UpdateSpatialMessageD(x, y, z, 0, cx, cy, cz, ox, oy, oz, mx, my, mz, dOffset, dRangeTerm, dSO1F2Message, dDualMessage);

		__syncthreads();

		cu_UpdateSpatialMessageD(x, y, z, 2, cx, cy, cz, ox, oy, oz, mx, my, mz, dOffset, dRangeTerm, dSO1F2Message, dDualMessage);

		__syncthreads();

		cu_UpdateSpatialMessageD(x, y, z, 4, cx, cy, cz, ox, oy, oz, mx, my, mz, dOffset, dRangeTerm, dSO1F2Message, dDualMessage);
	} else {
		cu_UpdateSpatialMessageD(x, y, z, 1, cx, cy, cz, ox, oy, oz, mx, my, mz, dOffset, dRangeTerm, dSO1F2Message, dDualMessage);

		__syncthreads();

		cu_UpdateSpatialMessageD(x, y, z, 3, cx, cy, cz, ox, oy, oz, mx, my, mz, dOffset, dRangeTerm, dSO1F2Message, dDualMessage);

		__syncthreads();

		cu_UpdateSpatialMessageD(x, y, z, 5, cx, cy, cz, ox, oy, oz, mx, my, mz, dOffset, dRangeTerm, dSO1F2Message, dDualMessage);
	}
}

__global__ static void cu_UpdateSpatialMessage_O2F3(unsigned int blocksInY, float invBlocksInY, int dir1, int ox, int oy, int oz, int mx, int my, int mz,
	REALV* dRangeTerm, REALV** dSO2, REALV** dSO2F3Message, REALV* dDualMessage)
{
	int cx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
#ifdef CU_USE_3D_BLOCK
	int cy = __umul24(blockIdx.y , blockDim.y) + threadIdx.y;
	int cz = __umul24(blockIdx.z , blockDim.z) + threadIdx.z;
#else
	int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
	int blockIdxy = blockIdx.y - __umul24(blockIdxz, blocksInY);
	int cy = __umul24(blockIdxy , blockDim.y) + threadIdx.y;
	int cz = __umul24(blockIdxz , blockDim.z) + threadIdx.z;
#endif
	int x1, y1, z1;
	size_t idx0, idx1, idx2;
	//
	extern __shared__ REALV sbuf[];

	x1 = cx + ox;
	y1 = cy + oy;
	z1 = cz + oz;

	int x0, y0, z0, dir0;
	int x2, y2, z2, dir2;
	REALV *m_fx0, *m_fx1, *m_fx2;
	REALV *m_fx0_u, *m_fx1_u, *m_fx2_u;
	REALV Y0[MAX_L8_1], Y1[MAX_L8_1], Y2[MAX_L8_1]; //Yf[MAX_L8_1]
	REALV *D;
	int y, k0, k1, k2, K, L, L2, L5, L6, L7, L8, L8_1; //yy
	REALV delta_0, delta_1, delta_2, v_fx;

	K = c_K;
	L = c_L;
	L2 = c_L2;
	L5 = c_L5;
	L6 = c_L6;
	L7 = c_L7;
	L8 = c_L8;
	L8_1 = c_L8_1;

	// eliminate impossible messages
	if (dir1 == 1 && (x1 <= 0 || x1 >= c_mesh_x-1)) { return; }
	if (dir1 == 4 && (y1 <= 0 || y1 >= c_mesh_y-1)) { return; }
	if (dir1 == 7 && (z1 <= 0 || z1 >= c_mesh_z-1)) { return; }

	switch (dir1) {
	case 1: 
		x0 = x1-1; y0 = y1  ; z0 = z1  ; dir0 = 0;
		x2 = x1+1; y2 = y1  ; z2 = z1  ; dir2 = 2;
		//
		idx0 = ((cz  )*my + (cy  ))*mx + (cx-1);
		idx1 = ((cz  )*my + (cy  ))*mx + (cx  );
		idx2 = ((cz  )*my + (cy  ))*mx + (cx+1);
		//
		D = &dSO2[0][idx1*L8_1];
		break;
	case 4:
		x0 = x1  ; y0 = y1-1; z0 = z1  ; dir0 = 3;
		x2 = x1  ; y2 = y1+1; z2 = z1  ; dir2 = 5;
		//
		idx0 = ((cz  )*my + (cy-1))*mx + (cx  );
		idx1 = ((cz  )*my + (cy  ))*mx + (cx  );
		idx2 = ((cz  )*my + (cy+1))*mx + (cx  );
		//
		D = &dSO2[1][idx1*L8_1];
		break;
	case 7:
		x0 = x1  ; y0 = y1  ; z0 = z1-1; dir0 = 6;
		x2 = x1  ; y2 = y1  ; z2 = z1+1; dir2 = 8;
		//
		idx0 = ((cz-1)*my + (cy  ))*mx + (cx  );
		idx1 = ((cz  )*my + (cy  ))*mx + (cx  );
		idx2 = ((cz+1)*my + (cy  ))*mx + (cx  );
		//
		D = &dSO2[2][idx1*L8_1];
		break;
	}

	int tidx = threadIdx.y * blockDim.x + threadIdx.x;
	int tsize_e = 3*K;

	m_fx0_u = &sbuf[tidx*tsize_e + 0*K];
	m_fx1_u = &sbuf[tidx*tsize_e + 1*K];
	m_fx2_u = &sbuf[tidx*tsize_e + 2*K];

	cu_Add2MessageSpatial_O2F3(m_fx0_u, x0, y0, z0, dir0, K, idx0, dRangeTerm, NULL, dSO2F3Message, dDualMessage);
	cu_Add2MessageSpatial_O2F3(m_fx1_u, x1, y1, z1, dir1, K, idx1, dRangeTerm, NULL, dSO2F3Message, dDualMessage);
	cu_Add2MessageSpatial_O2F3(m_fx2_u, x2, y2, z2, dir2, K, idx2, dRangeTerm, NULL, dSO2F3Message, dDualMessage);

	__syncthreads();

	m_fx0 = &dSO2F3Message[dir0][idx0*K];
	m_fx1 = &dSO2F3Message[dir1][idx1*K];
	m_fx2 = &dSO2F3Message[dir2][idx2*K];

	//////////////////////////////////////////
	for (y = 0; y <= L8; y++) {
		Y0[y] = INFINITE_S;
		Y1[y] = INFINITE_S;
		Y2[y] = INFINITE_S;
	}

	// make y = L4 when k0 = L, k1 = L, k2 = L

	// y = -2*x1 + x2 = -2*(k1-L) + (k2-L) = 2*k1 - k2 + L + (4*L)
	// y = [L, L7]
	for (k2 = 0; k2 < K; k2++) {
		for (k1 = 0; k1 < K; k1++) {
			y = -2*k1 + k2 + L5;
			delta_0 = m_fx1_u[k1] + m_fx2_u[k2];
			TRUNCATE(Y0[y], delta_0);
		}
	}
	// y = x0 + x2 = (k0-L) + (k2-L) = k0 + k2 - 2*L + (4*L)
	// y = [L2, L6]
	for (k2 = 0; k2 < K; k2++) {
		for (k0 = 0; k0 < K; k0++) {
			y = k0 + k2 + L2;
			delta_1 = m_fx0_u[k0] + m_fx2_u[k2];
			TRUNCATE(Y1[y], delta_1);
		}
	}
	// y = x0 - 2*x1 = (k0-L) - 2*(k1-L) = k0 - 2*k1 + L + (4*L)
	// y = [L, L7]
	for (k1 = 0; k1 < K; k1++) {
		for (k0 = 0; k0 < K; k0++) {
			y = k0 - 2*k1 + L5;
			delta_2 = m_fx0_u[k0] + m_fx1_u[k1];
			TRUNCATE(Y2[y], delta_2);
		}
	}
	//////////////////////////////////////////

	//////////////////////////////////////////
	// Calculating messages
	//////////////////////////////////////////
	for (k0 = 0; k0 < K; k0++) {
		delta_0 = D[L+k0-L] + Y0[L];
		for (y = L; y <= L7; y++) {
			v_fx = D[y+k0-L] + Y0[y];
			TRUNCATE(delta_0, v_fx);
		}
		m_fx0_u[k0] = delta_0;
	}
	cu_SubtractMin(m_fx0_u, m_fx0, K, delta_0);
	//////////////////////////////////////////
	for (k1 = 0; k1 < K; k1++) {
		delta_1 = D[L2-2*k1+L2] + Y1[L2];
		for (y = L2; y <= L6; y++) {
			v_fx = D[y-2*k1+L2] + Y1[y];
			TRUNCATE(delta_1, v_fx);
		}
		m_fx1_u[k1] = delta_1;
	}
	cu_SubtractMin(m_fx1_u, m_fx1, K, delta_1);
	//////////////////////////////////////////
	for (k2 = 0; k2 < K; k2++) {
		delta_2 = D[L+k2-L] + Y2[L];
		for (y = L; y <= L7; y++) {
			v_fx = D[y+k2-L] + Y2[y];
			TRUNCATE(delta_2, v_fx);
		}
		m_fx2_u[k2] = delta_2;
	}
	cu_SubtractMin(m_fx2_u, m_fx2, K, delta_2);
	//////////////////////////////////////////
}

__global__ static void cu_UpdateDualMessage(unsigned int blocksInY, float invBlocksInY,
	int ox, int oy, int oz, int mx, int my, int mz,
	REALV* ddcv, REALV** dRangeTerm, REALV*** dSO1F2Message, REALV*** dSO2F3Message, REALV** dDualMessage)
{
	int cx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
#ifdef CU_USE_3D_BLOCK
	int cy = __umul24(blockIdx.y , blockDim.y) + threadIdx.y;
	int cz = __umul24(blockIdx.z , blockDim.z) + threadIdx.z;
#else
	int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
	int blockIdxy = blockIdx.y - __umul24(blockIdxz, blocksInY);
	int cy = __umul24(blockIdxy , blockDim.y) + threadIdx.y;
	int cz = __umul24(blockIdxz , blockDim.z) + threadIdx.z;
#endif
	int x, y, z;
	size_t cidx, idx;
	//
	REALV *m_fx0_b, *m_fx1_b, *m_fx2_b;
	REALV m_fx0_u[MAX_K], m_fx1_u[MAX_K], m_fx2_u[MAX_K];
	REALV *m_fx0, *m_fx1, *m_fx2;
	REALV v_fx, delta_0, delta_1, delta_2;
	int k0, k1, k2, kk, k2_K_2, k1_K, K, K_2;
	//
	extern __shared__ REALV sbuf[];

	x = cx + ox;
	y = cy + oy;
	z = cz + oz;

	idx  = (z*c_mesh_y + y)*c_mesh_x + x;
	cidx = (cz*my + cy)*mx + cx;

	K = c_K;
	K_2 = c_K * c_K;

	int tidx = threadIdx.y * blockDim.x + threadIdx.x;
	int tsize_e = 3*K;
	m_fx0_b = &sbuf[tidx*tsize_e + 0*K];
	m_fx1_b = &sbuf[tidx*tsize_e + 1*K];
	m_fx2_b = &sbuf[tidx*tsize_e + 2*K];

	//////////////////////////////////////////
	//////////////////////////////////////////
	cu_Add2MessageDual(m_fx0_b, x, y, z, K, idx, dRangeTerm[0], dSO1F2Message[0], dSO2F3Message[0]);
	cu_Add2MessageDual(m_fx1_b, x, y, z, K, idx, dRangeTerm[1], dSO1F2Message[1], dSO2F3Message[1]);
	cu_Add2MessageDual(m_fx2_b, x, y, z, K, idx, dRangeTerm[2], dSO1F2Message[2], dSO2F3Message[2]);
	//////////////////////////////////////////
	//////////////////////////////////////////

	__syncthreads();

	//////////////////////////////////////////
	//////////////////////////////////////////
#if 1
	REALV *Dm;
	Dm = &ddcv[cidx*c_num_d];
#else
	REALV *D;
	REALV *Dm;
	D = &ddcv[idx*num_d];
	Dm = &sbuf[tidx*tsize_e + 6*K];
	memcpy(Dm, D, num_d * sizeof(REALV));
#endif
	m_fx0 = &dDualMessage[0][idx*K];
	m_fx1 = &dDualMessage[1][idx*K];
	m_fx2 = &dDualMessage[2][idx*K];

	for (k0 = 0; k0 < K; k0++) {
		delta_0 = INFINITE_S;
		for (k2 = 0; k2 < K; k2++) {
			kk = k2*K_2 + k0;
			for (k1 = 0; k1 < K; k1++) {
				v_fx = Dm[kk + k1*K] + m_fx1_b[k1] + m_fx2_b[k2];
				TRUNCATE(delta_0, v_fx);
			}
		}
		m_fx0_u[k0] = delta_0;
	}

	for (k1 = 0; k1 < K; k1++) {
		delta_1 = INFINITE_S;
		k1_K = k1 * K;
		for (k2 = 0; k2 < K; k2++) {
			kk = k2*K_2 + k1_K;
			for (k0 = 0; k0 < K; k0++) {
				v_fx = Dm[kk + k0] + m_fx0_b[k0] + m_fx2_b[k2];
				TRUNCATE(delta_1, v_fx);
			}
		}
		m_fx1_u[k1] = delta_1;
	}

	for (k2 = 0; k2 < K; k2++) {
		delta_2 = INFINITE_S;
		k2_K_2 = k2 * K_2;
		for (k1 = 0; k1 < K; k1++) {
			kk = k2_K_2 + k1 * K;
			for (k0 = 0; k0 < K; k0++) {
				v_fx = Dm[kk + k0] + m_fx0_b[k0] + m_fx1_b[k1];
				TRUNCATE(delta_2, v_fx);
			}
		}
		m_fx2_u[k2] = delta_2;
	}
	//////////////////////////////////////////
	//////////////////////////////////////////

	//////////////////////////////////////////
	//////////////////////////////////////////
	//if (plane != 0) {
		cu_SubtractMin(m_fx0_u, m_fx0, K, delta_0);
	//}
	//////////////////////////////////////////
	//if (plane != 1) {
		cu_SubtractMin(m_fx1_u, m_fx1, K, delta_1);
	//}
	//////////////////////////////////////////
	//if (plane != 2) {
		cu_SubtractMin(m_fx2_u, m_fx2, K, delta_2);
	//}
	//////////////////////////////////////////
	//////////////////////////////////////////
}

extern "C"
void cu_BP_S(int iter, REALV**** pdcv, REALV**** pOffset[3], REALV**** pRangeTerm[3], REALV**** pSO1[3][3], REALV**** pSO2[3][3],
	REALV**** pSO1F2Message[3][6], REALV**** pSO2F3Message[3][9], REALV**** pDualMessage[3], int iterPrev)
{
	int i, j, l, it;

#ifdef CU_USE_TIMER
#ifdef CU_USE_CUTIL
	unsigned int timer;
	cutCreateTimer(&timer);
	cutStartTimer(timer);
#else
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif
#endif

	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////
	if (iterPrev == 0) {
		if (mmode == 0) {
			cu_VolInit(ddcv, pdcv, 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, num_d);
		}
		for (i = 0; i < 3; i++) {
			cu_VolInit(hOffset[i], pOffset[i], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, 1);
			cu_VolInit(hRangeTerm[i], pRangeTerm[i], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, K);
			if (in_scv_w_O1F2 != -2) {
				if (mmode == 0) {
					for (j = 0; j < 3; j++) {
						cu_VolInit(hSO1[i][j], pSO1[i][j], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, L4_1);
					}
				}
				for (j = 0; j < 6; j++) {
					cu_VolInit(hSO1F2Message[i][j], pSO1F2Message[i][j], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, K);
				}
			}
			if (in_scv_w_O2F3 != -2) {
				if (mmode == 0) {
					for (j = 0; j < 3; j++) {
						cu_VolInit(hSO2[i][j], pSO2[i][j], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, L8_1);
					}
				}
				for (j = 0; j < 9; j++) {
					cu_VolInit(hSO2F3Message[i][j], pSO2F3Message[i][j], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, K);
				}
			}
			cu_VolInit(hDualMessage[i], pDualMessage[i], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, K);
		}
	}
	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////

#ifdef CU_USE_TIMER
#ifdef CU_USE_CUTIL
	cutStopTimer(timer);
	TRACE2("h->d time = %f\n", cutGetTimerValue(timer));
	cutDeleteTimer(timer);

	cutCreateTimer(&timer);
	cutStartTimer(timer);
#else
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	TRACE2("h->d time = %f\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif
#endif

	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////
	//cudaPrintfInit(blocksInX*blocksInY*blocksInZ*threadsInX*threadsInY*threadsInZ*256);

	for (it = 0; it < iter; it++) {
		///////////////////////////////////////////////////////////////////////////////////////
		if (in_scv_w_O1F2 != -2) {
			for (l = 0; l < 3; l++) {
				cu_UpdateSpatialMessage<<<Dg, Db, Ns>>>(blocksInY, invBlocksInY, 0, 0, 0, 0, mx, my, mz, hOffset[l], hRangeTerm[l], ddSO1F2Message[l], hDualMessage[l]);
				//
				//FILE* fp = fopen("culog.txt", "w");
				//cudaPrintfDisplay(fp, false);
				//fflush(fp);
				//fclose(fp);
			}
		} else if (in_scv_w_O2F3 != -2) {
			for (l = 0; l < 3; l++) {
				cu_UpdateSpatialMessage_O2F3<<<Dg, Db, Ns>>>(blocksInY, invBlocksInY, 1, 0, 0, 0, mx, my, mz, hRangeTerm[l], ddSO2[l], ddSO2F3Message[l], hDualMessage[l]);
				cu_UpdateSpatialMessage_O2F3<<<Dg, Db, Ns>>>(blocksInY, invBlocksInY, 4, 0, 0, 0, mx, my, mz, hRangeTerm[l], ddSO2[l], ddSO2F3Message[l], hDualMessage[l]);
				cu_UpdateSpatialMessage_O2F3<<<Dg, Db, Ns>>>(blocksInY, invBlocksInY, 7, 0, 0, 0, mx, my, mz, hRangeTerm[l], ddSO2[l], ddSO2F3Message[l], hDualMessage[l]);
			}
		}
		hipDeviceSynchronize();
		///////////////////////////////////////////////////////////////////////////////////////
		
		///////////////////////////////////////////////////////////////////////////////////////
		if (in_scv_w_O1F2 != -2) {
			for (l = 0; l < 3; l++) {
				cu_UpdateSpatialMessage<<<Dg, Db, Ns>>>(blocksInY, invBlocksInY, 1, 0, 0, 0, mx, my, mz, hOffset[l], hRangeTerm[l], ddSO1F2Message[l], hDualMessage[l]);
			}
		} else if (in_scv_w_O2F3 != -2) {
		}
		hipDeviceSynchronize();
		///////////////////////////////////////////////////////////////////////////////////////

		///////////////////////////////////////////////////////////////////////////////////////
		//*
		if (mmode == 0) {
			cu_UpdateDualMessage<<<Dg, Db, Ns>>>(blocksInY, invBlocksInY, 0, 0, 0, mx, my, mz, ddcv, dRangeTerm, dSO1F2Message, dSO2F3Message, dDualMessage);
		} else if (mmode == 1) {
			size_t msize_4 = msize / 4;
			int mz_4  =     mz / 4;
			int mz_42 = 2 * mz / 4;
			int mz_43 = 3 * mz / 4;
			//
			#if 0
			{
				size_t free_mem, total_mem, req_mem;
				CUDA_SAFE_CALL(hipMemGetInfo(&free_mem, &total_mem));
				req_mem = msize_4 * num_d * sizeof(REALV);
				if (req_mem > free_mem) {
					TRACE("req_mem = %u, free_mem = %u\n", req_mem, free_mem);
					return;
				}
			}
			#endif
			//
			CUDA_SAFE_CALL(hipMalloc(&ddcv, msize_4 * num_d * sizeof(REALV)));
			//
			cu_VolInit(ddcv, pdcv, 0, 0, 0    , mx, my, mz_4, mesh_x, mesh_y, mesh_z, num_d);
			cu_UpdateDualMessage<<<Dg_4, Db, Ns>>>(blocksInY, invBlocksInY, 0, 0, 0    , mx, my, mz_4, ddcv, dRangeTerm, dSO1F2Message, dSO2F3Message, dDualMessage);
			//
			cu_VolInit(ddcv, pdcv, 0, 0, mz_4 , mx, my, mz_4, mesh_x, mesh_y, mesh_z, num_d);
			cu_UpdateDualMessage<<<Dg_4, Db, Ns>>>(blocksInY, invBlocksInY, 0, 0, mz_4 , mx, my, mz_4, ddcv, dRangeTerm, dSO1F2Message, dSO2F3Message, dDualMessage);
			//
			cu_VolInit(ddcv, pdcv, 0, 0, mz_42, mx, my, mz_4, mesh_x, mesh_y, mesh_z, num_d);
			cu_UpdateDualMessage<<<Dg_4, Db, Ns>>>(blocksInY, invBlocksInY, 0, 0, mz_42, mx, my, mz_4, ddcv, dRangeTerm, dSO1F2Message, dSO2F3Message, dDualMessage);
			//
			cu_VolInit(ddcv, pdcv, 0, 0, mz_43, mx, my, mz_4, mesh_x, mesh_y, mesh_z, num_d);
			cu_UpdateDualMessage<<<Dg_4, Db, Ns>>>(blocksInY, invBlocksInY, 0, 0, mz_43, mx, my, mz_4, ddcv, dRangeTerm, dSO1F2Message, dSO2F3Message, dDualMessage);
			//
			hipFree(ddcv);
		}
		//*/
		hipDeviceSynchronize();
		///////////////////////////////////////////////////////////////////////////////////////

		if (it < iter-1) {
			TRACE2("iter %03d\n", it+iterPrev);
		}
	}

	//cudaPrintfEnd();
	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////

#ifdef CU_USE_TIMER
#ifdef CU_USE_CUTIL
	cutStopTimer(timer);
	TRACE2("avg time = %f\n", cutGetTimerValue(timer) / iter);
	cutDeleteTimer(timer);

	cutCreateTimer(&timer);
	cutStartTimer(timer);
#else
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	TRACE2("avg time = %f\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif
#endif

	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////
	for (i = 0; i < 3; i++) {
		if (in_scv_w_O1F2 != -2) {
			for (j = 0; j < 6; j++) {
				cu_VolCopy(hSO1F2Message[i][j], pSO1F2Message[i][j], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, K);
			}
		}
		if (in_scv_w_O2F3 != -2) {
			for (j = 0; j < 9; j++) {
				cu_VolCopy(hSO2F3Message[i][j], pSO2F3Message[i][j], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, K);
			}
		}
		cu_VolCopy(hDualMessage[i], pDualMessage[i], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, K);
	}
	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////

#ifdef CU_USE_TIMER
#ifdef CU_USE_CUTIL
	cutStopTimer(timer);
	TRACE2("d->h time = %f\n", cutGetTimerValue(timer));
	cutDeleteTimer(timer);
#else
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	TRACE2("d->h time = %f\n", elapsedTime);
   	hipEventDestroy(start);
	hipEventDestroy(stop); 
#endif
#endif
}
///////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////


///////////////////////////////////////////////////////////////////////////////////////
// Update Message for TRW_S
///////////////////////////////////////////////////////////////////////////////////////
__global__ static void cu_UpdateMessage_TRW_S_FW_O1F2(unsigned int blocksInY, float invBlocksInY, int ox, int oy, int oz, int mx, int my, int mz,
	REALV* dRangeTerm, REALV** dSO1F2Message, REALV* dDualMessage)
{
	int cx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
#ifdef CU_USE_3D_BLOCK
	int cy = __umul24(blockIdx.y , blockDim.y) + threadIdx.y;
	int cz = __umul24(blockIdx.z , blockDim.z) + threadIdx.z;
#else
	int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
	int blockIdxy = blockIdx.y - __umul24(blockIdxz, blocksInY);
	int cy = __umul24(blockIdxy , blockDim.y) + threadIdx.y;
	int cz = __umul24(blockIdxz , blockDim.z) + threadIdx.z;
#endif
	int x, y, z;
	int K;
	size_t idx, idxK;
	//
	extern __shared__ REALV sbuf[];

	x = cx + ox;
	y = cy + oy;
	z = cz + oz;

	idx  = (cz*my + cy)*mx + cx;
	K = c_K;
	idxK = idx * K;

	int i, k;
	BOOL update_s[6];
	REALV *m_fx;
	REALV *m_fx_b;
	REALV *m_fx_u;
	int ns;
	REALV r, vMin;

	m_fx_b = &sbuf[(threadIdx.y * blockDim.x + threadIdx.x)*2*K + 0*K];
	m_fx_u = &sbuf[(threadIdx.y * blockDim.x + threadIdx.x)*2*K + 1*K];

	// add the range term
	memcpy(m_fx_b, &dRangeTerm[idxK], K * sizeof(REALV));

	// add spatial messages
	for (i = 0; i < 6; i++) {
		update_s[i] = TRUE;
	}
	ns = 7;
	if (x > 0) {
		cu_Add2Message(m_fx_b, &dSO1F2Message[0][idxK], K);
	} else {
		update_s[0] = FALSE; ns--;
	}
	if (x < c_mesh_x-1) {
		cu_Add2Message(m_fx_b, &dSO1F2Message[1][idxK], K);
	} else {
		update_s[1] = FALSE; ns--;
	}
	if (y > 0) {
		cu_Add2Message(m_fx_b, &dSO1F2Message[2][idxK], K);
	} else {
		update_s[2] = FALSE; ns--;
	}
	if (y < c_mesh_y-1) {
		cu_Add2Message(m_fx_b, &dSO1F2Message[3][idxK], K);
	} else {
		update_s[3] = FALSE; ns--;
	}
	if (z > 0) {
		cu_Add2Message(m_fx_b, &dSO1F2Message[4][idxK], K);
	} else {
		update_s[4] = FALSE; ns--;
	}
	if (z < c_mesh_z-1) {
		cu_Add2Message(m_fx_b, &dSO1F2Message[5][idxK], K);
	} else {
		update_s[5] = FALSE; ns--;
	}
	r = 1.0 / ns;

	cu_Add2Message(m_fx_b, &dDualMessage[idxK], K);

	for (i = 0; i < 6; i++) {
		if (!update_s[i]) { continue; }

		m_fx = &dSO1F2Message[i][idxK];

		for (k = 0; k < K; k++) {
			m_fx_u[k] = r * m_fx_b[k] - m_fx[k];
		}
		vMin = m_fx_u[0];
		for (k = 1; k < K; k++) {
			TRUNCATE(vMin, m_fx_u[k]);
		}
		for (k = 0; k < K; k++) {
			m_fx[k] = m_fx_u[k] - vMin;
		}
	}
	{
		m_fx = &dDualMessage[idxK];

		for (k = 0; k < K; k++) {
			m_fx_u[k] = r * m_fx_b[k] - m_fx[k];
		}
		vMin = m_fx_u[0];
		for (k = 1; k < K; k++) {
			TRUNCATE(vMin, m_fx_u[k]);
		}
		for (k = 0; k < K; k++) {
			m_fx[k] = m_fx_u[k] - vMin;
		}
	}
}

__global__ static void cu_UpdateMessage_TRW_S_FW_O2F3(unsigned int blocksInY, float invBlocksInY, int ox, int oy, int oz, int mx, int my, int mz,
	REALV* dRangeTerm, REALV** dSO2F3Message, REALV* dDualMessage)
{
	int cx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
#ifdef CU_USE_3D_BLOCK
	int cy = __umul24(blockIdx.y , blockDim.y) + threadIdx.y;
	int cz = __umul24(blockIdx.z , blockDim.z) + threadIdx.z;
#else
	int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
	int blockIdxy = blockIdx.y - __umul24(blockIdxz, blocksInY);
	int cy = __umul24(blockIdxy , blockDim.y) + threadIdx.y;
	int cz = __umul24(blockIdxz , blockDim.z) + threadIdx.z;
#endif
	int x, y, z;
	int K;
	size_t idx, idxK;
	//
	extern __shared__ REALV sbuf[];

	x = cx + ox;
	y = cy + oy;
	z = cz + oz;

	idx  = (cz*my + cy)*mx + cx;
	K = c_K;
	idxK = idx * K;

	int i, k;
	BOOL update_s[9];
	REALV *m_fx;
	REALV *m_fx_b;
	REALV *m_fx_u;
	int ns;
	REALV r, vMin;

	m_fx_b = &sbuf[(threadIdx.y * blockDim.x + threadIdx.x)*2*K + 0*K];
	m_fx_u = &sbuf[(threadIdx.y * blockDim.x + threadIdx.x)*2*K + 1*K];

	// add the range term
	memcpy(m_fx_b, &dRangeTerm[idxK], K * sizeof(REALV));

	// add spatial messages
	for (i = 0; i < 9; i++) {
		update_s[i] = TRUE;
	}
	ns = 10;
	if (x < c_mesh_x-2) {				// f+ -> x
		cu_Add2Message(m_fx_b, &dSO2F3Message[0][idxK], K);
	} else {
		update_s[0] = FALSE; ns--;
	}
	if (x > 0 && x < c_mesh_x-1) {	// f0 -> x
		cu_Add2Message(m_fx_b, &dSO2F3Message[1][idxK], K);
	} else {
		update_s[1] = FALSE; ns--;
	}
	if (x > 1) {					// f- -> x
		cu_Add2Message(m_fx_b, &dSO2F3Message[2][idxK], K);
	} else {
		update_s[2] = FALSE; ns--;
	}
	if (y < c_mesh_y-2) {
		cu_Add2Message(m_fx_b, &dSO2F3Message[3][idxK], K);
	} else {
		update_s[3] = FALSE; ns--;
	}
	if (y > 0 && y < c_mesh_y-1) {
		cu_Add2Message(m_fx_b, &dSO2F3Message[4][idxK], K);
	} else {
		update_s[4] = FALSE; ns--;
	}
	if (y > 1) {
		cu_Add2Message(m_fx_b, &dSO2F3Message[5][idxK], K);
	} else {
		update_s[5] = FALSE; ns--;
	}
	if (z < c_mesh_z-2) {
		cu_Add2Message(m_fx_b, &dSO2F3Message[6][idxK], K);
	} else {
		update_s[6] = FALSE; ns--;
	}
	if (z > 0 && z < c_mesh_z-1) {
		cu_Add2Message(m_fx_b, &dSO2F3Message[7][idxK], K);
	} else {
		update_s[7] = FALSE; ns--;
	}
	if (z > 1) {
		cu_Add2Message(m_fx_b, &dSO2F3Message[8][idxK], K);
	} else {
		update_s[8] = FALSE; ns--;
	}

	r = 1.0 / ns;

	cu_Add2Message(m_fx_b, &dDualMessage[idxK], K);

	for (i = 0; i < 9; i++) {
		if (!update_s[i]) { continue; }

		m_fx = &dSO2F3Message[i][idxK];

		for (k = 0; k < K; k++) {
			m_fx_u[k] = r * m_fx_b[k] - m_fx[k];
		}
		vMin = m_fx_u[0];
		for (k = 1; k < K; k++) {
			TRUNCATE(vMin, m_fx_u[k]);
		}
		for (k = 0; k < K; k++) {
			m_fx[k] = m_fx_u[k] - vMin;
		}
	}
	{
		m_fx = &dDualMessage[idxK];

		for (k = 0; k < K; k++) {
			m_fx_u[k] = r * m_fx_b[k] - m_fx[k];
		}
		vMin = m_fx_u[0];
		for (k = 1; k < K; k++) {
			TRUNCATE(vMin, m_fx_u[k]);
		}
		for (k = 0; k < K; k++) {
			m_fx[k] = m_fx_u[k] - vMin;
		}
	}
}

__global__ static void cu_UpdateMessage_TRW_S_FW_O1F2_O2F3(unsigned int blocksInY, float invBlocksInY, int ox, int oy, int oz, int mx, int my, int mz,
	REALV* dRangeTerm, REALV** dSO1F2Message, REALV** dSO2F3Message, REALV* dDualMessage)
{
	int cx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
#ifdef CU_USE_3D_BLOCK
	int cy = __umul24(blockIdx.y , blockDim.y) + threadIdx.y;
	int cz = __umul24(blockIdx.z , blockDim.z) + threadIdx.z;
#else
	int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
	int blockIdxy = blockIdx.y - __umul24(blockIdxz, blocksInY);
	int cy = __umul24(blockIdxy , blockDim.y) + threadIdx.y;
	int cz = __umul24(blockIdxz , blockDim.z) + threadIdx.z;
#endif
	int x, y, z;
	int K;
	size_t idx, idxK;
	//
	extern __shared__ REALV sbuf[];

	x = cx + ox;
	y = cy + oy;
	z = cz + oz;

	idx  = (cz*my + cy)*mx + cx;
	K = c_K;
	idxK = idx * K;

	int i, k;
	BOOL update_s_O1[9];
	BOOL update_s_O2[9];
	REALV *m_fx;
	REALV *m_fx_b;
	REALV *m_fx_u;
	int ns_O1;
	int ns_O2;
	REALV r, vMin;

	m_fx_b = &sbuf[(threadIdx.y * blockDim.x + threadIdx.x)*2*K + 0*K];
	m_fx_u = &sbuf[(threadIdx.y * blockDim.x + threadIdx.x)*2*K + 1*K];

	// add the range term
	memcpy(m_fx_b, &dRangeTerm[idxK], K * sizeof(REALV));

	// add spatial messages for O1
	for (i = 0; i < 6; i++) {
		update_s_O1[i] = TRUE;
	}
	ns_O1 = 6;
	if (x > 0) {
		cu_Add2Message(m_fx_b, &dSO1F2Message[0][idxK], K);
	} else {
		update_s_O1[0] = FALSE; ns_O1--;
	}
	if (x < c_mesh_x-1) {
		cu_Add2Message(m_fx_b, &dSO1F2Message[1][idxK], K);
	} else {
		update_s_O1[1] = FALSE; ns_O1--;
	}
	if (y > 0) {
		cu_Add2Message(m_fx_b, &dSO1F2Message[2][idxK], K);
	} else {
		update_s_O1[2] = FALSE; ns_O1--;
	}
	if (y < c_mesh_y-1) {
		cu_Add2Message(m_fx_b, &dSO1F2Message[3][idxK], K);
	} else {
		update_s_O1[3] = FALSE; ns_O1--;
	}
	if (z > 0) {
		cu_Add2Message(m_fx_b, &dSO1F2Message[4][idxK], K);
	} else {
		update_s_O1[4] = FALSE; ns_O1--;
	}
	if (z < c_mesh_z-1) {
		cu_Add2Message(m_fx_b, &dSO1F2Message[5][idxK], K);
	} else {
		update_s_O1[5] = FALSE; ns_O1--;
	}

	// add spatial messages for O2
	for (i = 0; i < 9; i++) {
		update_s_O2[i] = TRUE;
	}
	ns_O2 = 9;
	if (x < c_mesh_x-2) {				// f+ -> x
		cu_Add2Message(m_fx_b, &dSO2F3Message[0][idxK], K);
	} else {
		update_s_O2[0] = FALSE; ns_O2--;
	}
	if (x > 0 && x < c_mesh_x-1) {	// f0 -> x
		cu_Add2Message(m_fx_b, &dSO2F3Message[1][idxK], K);
	} else {
		update_s_O2[1] = FALSE; ns_O2--;
	}
	if (x > 1) {					// f- -> x
		cu_Add2Message(m_fx_b, &dSO2F3Message[2][idxK], K);
	} else {
		update_s_O2[2] = FALSE; ns_O2--;
	}
	if (y < c_mesh_y-2) {
		cu_Add2Message(m_fx_b, &dSO2F3Message[3][idxK], K);
	} else {
		update_s_O2[3] = FALSE; ns_O2--;
	}
	if (y > 0 && y < c_mesh_y-1) {
		cu_Add2Message(m_fx_b, &dSO2F3Message[4][idxK], K);
	} else {
		update_s_O2[4] = FALSE; ns_O2--;
	}
	if (y > 1) {
		cu_Add2Message(m_fx_b, &dSO2F3Message[5][idxK], K);
	} else {
		update_s_O2[5] = FALSE; ns_O2--;
	}
	if (z < c_mesh_z-2) {
		cu_Add2Message(m_fx_b, &dSO2F3Message[6][idxK], K);
	} else {
		update_s_O2[6] = FALSE; ns_O2--;
	}
	if (z > 0 && z < c_mesh_z-1) {
		cu_Add2Message(m_fx_b, &dSO2F3Message[7][idxK], K);
	} else {
		update_s_O2[7] = FALSE; ns_O2--;
	}
	if (z > 1) {
		cu_Add2Message(m_fx_b, &dSO2F3Message[8][idxK], K);
	} else {
		update_s_O2[8] = FALSE; ns_O2--;
	}

	r = 1.0 / (ns_O1 + ns_O2 + 1);

	cu_Add2Message(m_fx_b, &dDualMessage[idxK], K);

	for (i = 0; i < 6; i++) {
		if (!update_s_O1[i]) { continue; }

		m_fx = &dSO1F2Message[i][idxK];

		for (k = 0; k < K; k++) {
			m_fx_u[k] = r * m_fx_b[k] - m_fx[k];
		}
		vMin = m_fx_u[0];
		for (k = 1; k < K; k++) {
			TRUNCATE(vMin, m_fx_u[k]);
		}
		for (k = 0; k < K; k++) {
			m_fx[k] = m_fx_u[k] - vMin;
		}
	}
	for (i = 0; i < 9; i++) {
		if (!update_s_O2[i]) { continue; }

		m_fx = &dSO2F3Message[i][idxK];

		for (k = 0; k < K; k++) {
			m_fx_u[k] = r * m_fx_b[k] - m_fx[k];
		}
		vMin = m_fx_u[0];
		for (k = 1; k < K; k++) {
			TRUNCATE(vMin, m_fx_u[k]);
		}
		for (k = 0; k < K; k++) {
			m_fx[k] = m_fx_u[k] - vMin;
		}
	}
	{
		m_fx = &dDualMessage[idxK];

		for (k = 0; k < K; k++) {
			m_fx_u[k] = r * m_fx_b[k] - m_fx[k];
		}
		vMin = m_fx_u[0];
		for (k = 1; k < K; k++) {
			TRUNCATE(vMin, m_fx_u[k]);
		}
		for (k = 0; k < K; k++) {
			m_fx[k] = m_fx_u[k] - vMin;
		}
	}
}

__global__ static void cu_UpdateMessage_TRW_S_BW(unsigned int blocksInY, float invBlocksInY, int ox, int oy, int oz, int mx, int my, int mz,
	REALV* dRangeTerm, REALV** dSO1F2Message, REALV** dSO2F3Message, REALV* dDualMessage)
{
	int cx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
#ifdef CU_USE_3D_BLOCK
	int cy = __umul24(blockIdx.y , blockDim.y) + threadIdx.y;
	int cz = __umul24(blockIdx.z , blockDim.z) + threadIdx.z;
#else
	int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
	int blockIdxy = blockIdx.y - __umul24(blockIdxz, blocksInY);
	int cy = __umul24(blockIdxy , blockDim.y) + threadIdx.y;
	int cz = __umul24(blockIdxz , blockDim.z) + threadIdx.z;
#endif
	int x, y, z;
	int K;
	size_t idx;
	//
	extern __shared__ REALV sbuf[];

	x = cx + ox;
	y = cy + oy;
	z = cz + oz;

	idx  = (cz*my + cy)*mx + cx;
	K = c_K;

	int k;
	REALV *m_fx_b;
	REALV vMin;

	m_fx_b = &sbuf[(threadIdx.y * blockDim.x + threadIdx.x)*K];

	cu_Add2MessageDual(m_fx_b, x, y, z, K, idx, dRangeTerm, dSO1F2Message, dSO2F3Message);

	cu_Add2Message(m_fx_b, &dDualMessage[idx*K], K);

	vMin = m_fx_b[0];
	for (k = 1; k < K; k++) {
		TRUNCATE(vMin, m_fx_b[k]);
	}

	//dLowerBound += vMin;
	//atomicAdd(&dLowerBound, 100);
	atomicAdd(&dLowerBound, (int)(vMin));

	//cuPrintf("\tbx = %3d, by = %3d, bz = %3d, tx = %3d, ty = %3d, tz = %3d, idx = (%3d, %3d, %3d), %d\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z, cx, cy, cz, dLowerBound);
}

__global__ static void cu_UpdateSpatialMessage_TRW_S_BW_O1F2(unsigned int blocksInY, float invBlocksInY, int direction, int ox, int oy, int oz, int mx, int my, int mz,
	REALV* dOffset, REALV** dSO1, REALV** dSO1F2Message)
{
	int cx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
#ifdef CU_USE_3D_BLOCK
	int cy = __umul24(blockIdx.y , blockDim.y) + threadIdx.y;
	int cz = __umul24(blockIdx.z , blockDim.z) + threadIdx.z;
#else
	int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
	int blockIdxy = blockIdx.y - __umul24(blockIdxz, blocksInY);
	int cy = __umul24(blockIdxy , blockDim.y) + threadIdx.y;
	int cz = __umul24(blockIdxz , blockDim.z) + threadIdx.z;
#endif
	int x, y, z;
	int K;
	size_t idx0, idx1;
	//
	extern __shared__ REALV sbuf[];

	x = cx + ox;
	y = cy + oy;
	z = cz + oz;

	K = c_K;

	int direction1;
	int x1, y1, z1;
	REALV *m_fx0, *m_fx1;
	REALV *m_buf;
	REALV delta_f, delta_0, delta_1;
	REALV* c_disp_e;

	x1 = x; y1 = y; z1 = z; // destination
	if (direction == 0) {
		if (x == c_mesh_x-1) { return; }
		x1++;
		c_disp_e = c_disp_ex;
		direction1 = 1;
	} else if (direction == 1) {
		if (x == 0       ) { return; }
		x1--;
		c_disp_e = c_disp_ex;
		direction1 = 0;
	} else if (direction == 2) {
		if (y == c_mesh_y-1) { return; }
		y1++;
		c_disp_e = c_disp_ey;
		direction1 = 3;
	} else if (direction == 3) {
		if (y == 0       ) { return; }
		y1--;
		c_disp_e = c_disp_ey;
		direction1 = 2;
	} else if (direction == 4) {
		if (z == c_mesh_z-1) { return; }
		z1++;
		c_disp_e = c_disp_ez;
		direction1 = 5;
	} else if (direction == 5) {
		if (z == 0       ) { return; }
		z1--;
		c_disp_e = c_disp_ez;
		direction1 = 4;
	}

	idx0 = (z *c_mesh_y + y )*c_mesh_x + x ;
	idx1 = (z1*c_mesh_y + y1)*c_mesh_x + x1;

	//////////////////////////////////////////
	//////////////////////////////////////////
	REALV d0;
#if 0
	REALV s, T;
#endif
	REALV *Y0, *Y1;
	REALV r_fx0, r_fx1;
	int k0, k1;

	m_fx0 = &dSO1F2Message[direction1][idx0*K];
	m_fx1 = &dSO1F2Message[direction ][idx1*K];
	r_fx0 = r_fx1 = 0.5;
#ifdef O1_USE_OFFSET
	d0 = dOffset[idx0] - dOffset[idx1];
#else
	d0 = 0;
#endif
#if 0
	s = c_alpha_O1;
	T = c_d_O1;
#endif

	//////////////////////////////////////////
#if 0
	delta_f = INFINITE_S;
	for (k1 = 0; k1 < K; k1++) {
		for (k0 = 0; k0 < K; k0++) {
			v_fx = min(s * fabs(d0+disp_e[k0]-disp_e[k1]), T) + m_fx0[k0] + m_fx1[k1];
			TRUNCATE(delta_f, v_fx);
		}
	}
#else
	delta_f = 0;
#endif
	//////////////////////////////////////////

	//////////////////////////////////////////
	int tidx = threadIdx.y * blockDim.x + threadIdx.x;
	int tsize_e = 3*K;

	m_buf = &sbuf[tidx*tsize_e + 0*K];
	Y0    = &sbuf[tidx*tsize_e + 1*K];
	Y1    = &sbuf[tidx*tsize_e + 2*K];
	//////////////////////////////////////////

#if 1
	//////////////////////////////////////////
	cu_ComputeSpatialMessageDT(Y0, m_fx1, m_buf, x, y, z, d0, K, c_nL, c_disp_e);
	for (k0 = 0; k0 < K; k0++) {
		Y0[k0] = r_fx0 * (Y0[k0]-delta_f) + (r_fx0-1)*m_fx0[k0];
	}
	delta_0 = Y0[0];
	for (k0 = 1; k0 < K; k0++) { 
		TRUNCATE(delta_0, Y0[k0]);
	}
	//
	cu_ComputeSpatialMessageDT(Y1, m_fx0, m_buf, x1, y1, z1, -d0, K, c_nL, c_disp_e);
	for (k1 = 0; k1 < K; k1++) {
		Y1[k1] = r_fx1 * (Y1[k1]-delta_f) + (r_fx1-1)*m_fx1[k1];
	}
	delta_1 = Y1[0];
	for (k1 = 1; k1 < K; k1++) { 
		TRUNCATE(delta_1, Y1[k1]);
	}
	//////////////////////////////////////////
#else
	//////////////////////////////////////////
	for (k0 = 0; k0 < K; k0++) {
		delta_0 = min(s * fabs(-d0+disp_e[k0]-disp_e[0]), T) + m_fx1[0];
		for (k1 = 0; k1 < K; k1++) {
			v_fx = min(s * fabs(-d0+disp_e[k0]-disp_e[k1]), T) + m_fx1[k1];
			TRUNCATE(delta_0, v_fx);
		}
		Y0[k0] = r_fx0 * (delta_0-delta_f) + (r_fx0-1)*m_fx0[k0];
	}
	delta_0 = Y0[0];
	for (k0 = 1; k0 < K; k0++) { 
		TRUNCATE(delta_0, Y0[k0]);
	}
	//
	for (k1 = 0; k1 < K; k1++) {
		delta_1 = min(s * fabs(-d0+disp_e[0]-disp_e[k1]), T) + m_fx0[0];
		for (k0 = 0; k0 < K; k0++) {
			v_fx = min(s * fabs(-d0+disp_e[k0]-disp_e[k1]), T) + m_fx0[k0];
			TRUNCATE(delta_1, v_fx);
		}
		Y1[k1] = r_fx1 * (delta_1-delta_f) + (r_fx1-1)*m_fx1[k1];
	}
	delta_1 = Y1[0];
	for (k1 = 1; k1 < K; k1++) { 
		TRUNCATE(delta_1, Y1[k1]);
	}
	//////////////////////////////////////////
#endif

	//////////////////////////////////////////
	for (k0 = 0; k0 < K; k0++) { 
		m_fx0[k0] = Y0[k0] - delta_0;
	}
	for (k1 = 0; k1 < K; k1++) { 
		m_fx1[k1] = Y1[k1] - delta_1;
	}
	//////////////////////////////////////////
	
	//////////////////////////////////////////
	//lowerBound += delta_f + delta_0 + delta_1;
	atomicAdd(&dLowerBound, (int)(delta_f + delta_0 + delta_1));
	//////////////////////////////////////////

	//////////////////////////////////////////
	//////////////////////////////////////////
}

__global__ static void cu_UpdateSpatialMessage_TRW_S_BW_O2F3(unsigned int blocksInY, float invBlocksInY, int dir1, int ox, int oy, int oz, int mx, int my, int mz,
	REALV** dSO2, REALV** dSO2F3Message)
{
	int cx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
#ifdef CU_USE_3D_BLOCK
	int cy = __umul24(blockIdx.y , blockDim.y) + threadIdx.y;
	int cz = __umul24(blockIdx.z , blockDim.z) + threadIdx.z;
#else
	int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
	int blockIdxy = blockIdx.y - __umul24(blockIdxz, blocksInY);
	int cy = __umul24(blockIdxy , blockDim.y) + threadIdx.y;
	int cz = __umul24(blockIdxz , blockDim.z) + threadIdx.z;
#endif
	int x1, y1, z1;
	size_t idx0, idx1, idx2;
	//
	extern __shared__ REALV sbuf[];

	x1 = cx + ox;
	y1 = cy + oy;
	z1 = cz + oz;

#if 0
	int x0, y0, z0, dir0;
	int x2, y2, z2, dir2;
#else
	int dir0;
	int dir2;
#endif
	REALV *m_fx0, *m_fx1, *m_fx2;
	REALV *m_fx0_u, *m_fx1_u, *m_fx2_u;
	REALV Y0[MAX_L8_1], Y1[MAX_L8_1], Y2[MAX_L8_1], Yf[MAX_L8_1];
	REALV *D;
	int y, yy, k, k0, k1, k2, K, L, L2, L5, L6, L7, L8, L8_1;
	REALV delta_0, delta_1, delta_2, delta_f, v_fx;
	REALV r_fx0, r_fx1, r_fx2;

	K = c_K;
	L = c_L;
	L2 = c_L2;
	L5 = c_L5;
	L6 = c_L6;
	L7 = c_L7;
	L8 = c_L8;
	L8_1 = c_L8_1;

	// eliminate impossible messages
	if (dir1 == 1 && (x1 <= 0 || x1 >= c_mesh_x-1)) { return; }
	if (dir1 == 4 && (y1 <= 0 || y1 >= c_mesh_y-1)) { return; }
	if (dir1 == 7 && (z1 <= 0 || z1 >= c_mesh_z-1)) { return; }

	switch (dir1) {
	case 1: 
#if 0
		x0 = x1-1; y0 = y1  ; z0 = z1  ; dir0 = 0;
		x2 = x1+1; y2 = y1  ; z2 = z1  ; dir2 = 2;
#else
		dir0 = 0;
		dir2 = 2;
#endif
		//
		idx0 = ((cz  )*my + (cy  ))*mx + (cx-1);
		idx1 = ((cz  )*my + (cy  ))*mx + (cx  );
		idx2 = ((cz  )*my + (cy  ))*mx + (cx+1);
		//
		D = &dSO2[0][idx1*L8_1];
		break;
	case 4:
#if 0
		x0 = x1  ; y0 = y1-1; z0 = z1  ; dir0 = 3;
		x2 = x1  ; y2 = y1+1; z2 = z1  ; dir2 = 5;
#else
		dir0 = 3;
		dir2 = 5;
#endif
		//
		idx0 = ((cz  )*my + (cy-1))*mx + (cx  );
		idx1 = ((cz  )*my + (cy  ))*mx + (cx  );
		idx2 = ((cz  )*my + (cy+1))*mx + (cx  );
		//
		D = &dSO2[1][idx1*L8_1];
		break;
	case 7:
#if 0
		x0 = x1  ; y0 = y1  ; z0 = z1-1; dir0 = 6;
		x2 = x1  ; y2 = y1  ; z2 = z1+1; dir2 = 8;
#else
		dir0 = 6;
		dir2 = 8;
#endif
		//
		idx0 = ((cz-1)*my + (cy  ))*mx + (cx  );
		idx1 = ((cz  )*my + (cy  ))*mx + (cx  );
		idx2 = ((cz+1)*my + (cy  ))*mx + (cx  );
		//
		D = &dSO2[2][idx1*L8_1];
		break;
	}

	int tidx = threadIdx.y * blockDim.x + threadIdx.x;
	int tsize_e = 3*K;

	m_fx0_u = &sbuf[tidx*tsize_e + 0*K];
	m_fx1_u = &sbuf[tidx*tsize_e + 1*K];
	m_fx2_u = &sbuf[tidx*tsize_e + 2*K];

	m_fx0 = &dSO2F3Message[dir0][idx0*K];
	m_fx1 = &dSO2F3Message[dir1][idx1*K];
	m_fx2 = &dSO2F3Message[dir2][idx2*K];
	r_fx0 = r_fx1 = r_fx2 = 1.0 / 3.0;

	//////////////////////////////////////////
	for (y = 0; y <= L8; y++) {
		Y0[y] = INFINITE_S;
		Y1[y] = INFINITE_S;
		Y2[y] = INFINITE_S;
		Yf[y] = INFINITE_S;
	}

	memcpy(m_fx0_u, m_fx0, K * sizeof(REALV));
	memcpy(m_fx1_u, m_fx1, K * sizeof(REALV));
	memcpy(m_fx2_u, m_fx2, K * sizeof(REALV));

	// make y = L4 when k0 = L, k1 = L, k2 = L

	// y = -2*x1 + x2 = -2*(k1-L) + (k2-L) = 2*k1 - k2 + L + (4*L)
	// y = [L, L7]
	for (k2 = 0; k2 < K; k2++) {
		for (k1 = 0; k1 < K; k1++) {
			y = -2*k1 + k2 + L5;
			//delta_0 = m_fx1[k1] + m_fx2[k2];
			delta_0 = m_fx1_u[k1] + m_fx2_u[k2];
			TRUNCATE(Y0[y], delta_0);
		}
	}
	// y = x0 + x2 = (k0-L) + (k2-L) = k0 + k2 - 2*L + (4*L)
	// y = [L2, L6]
	for (k2 = 0; k2 < K; k2++) {
		for (k0 = 0; k0 < K; k0++) {
			y = k0 + k2 + L2;
			//delta_1 = m_fx0[k0] + m_fx2[k2];
			delta_1 = m_fx0_u[k0] + m_fx2_u[k2];
			TRUNCATE(Y1[y], delta_1);
		}
	}
	// y = x0 - 2*x1 = (k0-L) - 2*(k1-L) = k0 - 2*k1 + L + (4*L)
	// y = [L, L7]
	for (k1 = 0; k1 < K; k1++) {
		for (k0 = 0; k0 < K; k0++) {
			y = k0 - 2*k1 + L5;
			//delta_2 = m_fx0[k0] + m_fx1[k1];
			delta_2 = m_fx0_u[k0] + m_fx1_u[k1];
			TRUNCATE(Y2[y], delta_2);
		}
	}
	//
	// y = k0 - 2*k1 + k2 + (4*L) = k0 + y0 (= -2*k2 + k3 + 5*L) - L
	// y = [0, L8]
	for (k0 = 0; k0 < K; k0++) {
		for (y = L; y <= L7; y++) {
			yy = y + k0 - L;
			//delta_f = m_fx0[k0] + Y0[y];
			delta_f = m_fx0_u[k0] + Y0[y];
			TRUNCATE(Yf[yy], delta_f);
		}
	}
	//
    delta_f = Yf[0] + D[0];
	for (k = 1; k <= L8; k++) {
		Yf[k] += D[k];
		TRUNCATE(delta_f, Yf[k]);
	}
	//for (k = 0; k < L8_1; k++) { 
	//	Yf[k] -= delta_f;
	//}
	//////////////////////////////////////////

	//////////////////////////////////////////
	// Calculating messages
	//////////////////////////////////////////
	for (k0 = 0; k0 < K; k0++) {
		delta_0 = D[L+k0-L] + Y0[L];
		for (y = L; y <= L7; y++) {
			v_fx = D[y+k0-L] + Y0[y];
			TRUNCATE(delta_0, v_fx);
		}
		m_fx0_u[k0] = r_fx0 * (delta_0-delta_f) + (r_fx0-1)*m_fx0[k0];
	}
	cu_SubtractMin(m_fx0_u, m_fx0, K, delta_0);
	//////////////////////////////////////////
	for (k1 = 0; k1 < K; k1++) {
		delta_1 = D[L2-2*k1+L2] + Y1[L2];
		for (y = L2; y <= L6; y++) {
			v_fx = D[y-2*k1+L2] + Y1[y];
			TRUNCATE(delta_1, v_fx);
		}
		m_fx1_u[k1] = r_fx1 * (delta_1-delta_f) + (r_fx1-1)*m_fx1[k1];
	}
	cu_SubtractMin(m_fx1_u, m_fx1, K, delta_1);
	//////////////////////////////////////////
	for (k2 = 0; k2 < K; k2++) {
		delta_2 = D[L+k2-L] + Y2[L];
		for (y = L; y <= L7; y++) {
			v_fx = D[y+k2-L] + Y2[y];
			TRUNCATE(delta_2, v_fx);
		}
		m_fx2_u[k2] = r_fx2 * (delta_2-delta_f) + (r_fx2-1)*m_fx2[k2];
	}
	cu_SubtractMin(m_fx2_u, m_fx2, K, delta_2);
	//////////////////////////////////////////

	//////////////////////////////////////////
	// Updating lower bound
	//////////////////////////////////////////
	//lowerBound += delta_f + delta_0 + delta_1 + delta_2;
	atomicAdd(&dLowerBound, (int)(delta_f + delta_0 + delta_1 + delta_2));
	//////////////////////////////////////////
}

__global__ static void cu_UpdateDualMessage_TRW_S_BW(unsigned int blocksInY, float invBlocksInY,
	int ox, int oy, int oz, int mx, int my, int mz,
	REALV* ddcv, REALV** dDualMessage)
{
	int cx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
#ifdef CU_USE_3D_BLOCK
	int cy = __umul24(blockIdx.y , blockDim.y) + threadIdx.y;
	int cz = __umul24(blockIdx.z , blockDim.z) + threadIdx.z;
#else
	int blockIdxz = __float2uint_rd(blockIdx.y * invBlocksInY);
	int blockIdxy = blockIdx.y - __umul24(blockIdxz, blocksInY);
	int cy = __umul24(blockIdxy , blockDim.y) + threadIdx.y;
	int cz = __umul24(blockIdxz , blockDim.z) + threadIdx.z;
#endif
	int x, y, z;
	size_t cidx, idx;
	//
	REALV *m_fx0_u, *m_fx1_u, *m_fx2_u;
	REALV m_fx0_b[MAX_K], m_fx1_b[MAX_K], m_fx2_b[MAX_K], *m_fx_buf;
	REALV *m_fx0, *m_fx1, *m_fx2;
	REALV r_fx0, r_fx1, r_fx2;
	REALV *Dm;
	REALV m_fx2_k2, fxfx, v_fx, delta_f, delta_0, delta_1, delta_2;
	int k0, k1, k2, kk, k2_K_2, k1_K, K, K_2;
	//
	extern __shared__ REALV sbuf[];

	x = cx + ox;
	y = cy + oy;
	z = cz + oz;

	idx  = (z*c_mesh_y + y)*c_mesh_x + x;
	cidx = (cz*my + cy)*mx + cx;

	K = c_K;
	K_2 = c_K * c_K;

	int tidx = threadIdx.y * blockDim.x + threadIdx.x;
	int tsize_e = 4*K;
	m_fx0_u  = &sbuf[tidx*tsize_e + 0*K];
	m_fx1_u  = &sbuf[tidx*tsize_e + 1*K];
	m_fx2_u  = &sbuf[tidx*tsize_e + 2*K];
	m_fx_buf = &sbuf[tidx*tsize_e + 3*K];

	//////////////////////////////////////////
	//////////////////////////////////////////
	Dm = &ddcv[cidx*c_num_d];
	m_fx0 = &dDualMessage[0][idx*K];
	m_fx1 = &dDualMessage[1][idx*K];
	m_fx2 = &dDualMessage[2][idx*K];
	r_fx0 = r_fx1 = r_fx2 = 1.0 / 3.0;

	memcpy(m_fx0_u, m_fx0, K * sizeof(REALV));
	memcpy(m_fx1_u, m_fx1, K * sizeof(REALV));
	memcpy(m_fx2_u, m_fx2, K * sizeof(REALV));

	delta_f = INFINITE_S;
	for (k2 = 0; k2 < K; k2++) {
		k2_K_2 = k2*K_2;
		//m_fx2_k2 = m_fx2[k2];
		m_fx2_k2 = m_fx2_u[k2];
		for (k1 = 0; k1 < K; k1++) {
			kk = k2_K_2 + k1*K;
			//fxfx = m_fx1[k1] + m_fx2_k2;
			fxfx = m_fx1_u[k1] + m_fx2_k2;
			for (k0 = 0; k0 < K; k0++) {
				//v_fx = Dm[kk + k0] + m_fx0[k0] + fxfx;
				v_fx = Dm[kk + k0] + m_fx0_u[k0] + fxfx;
				TRUNCATE(delta_f, v_fx);
				//
				//Dt[kk + k0] = v_fx;
			}
		}
	}

	for (k0 = 0; k0 < K; k0++) {
		delta_0 = INFINITE_S;
		for (k2 = 0; k2 < K; k2++) {
			kk = k2*K_2 + k0;
			for (k1 = 0; k1 < K; k1++) {
				//v_fx = Dt[kk + k1*K];
				//v_fx = Dm[kk + k1*K] + m_fx1[k1] + m_fx2[k2];
				v_fx = Dm[kk + k1*K] + m_fx1_u[k1] + m_fx2_u[k2];
				TRUNCATE(delta_0, v_fx);
			}
		}
		//m_fx0_u[k0] = r_fx0 * (delta_0-delta_f) - m_fx0[k0];
		//m_fx0_u[k0] = r_fx0 * (delta_0-delta_f) + (r_fx0-1) * m_fx0[k0];
		m_fx_buf[k0] = r_fx0 * (delta_0-delta_f) + (r_fx0-1) * m_fx0_u[k0];
	}
	memcpy(m_fx0_b, m_fx_buf, K * sizeof(REALV));

	for (k1 = 0; k1 < K; k1++) {
		delta_1 = INFINITE_S;
		k1_K = k1 * K;
		for (k2 = 0; k2 < K; k2++) {
			kk = k2*K_2 + k1_K;
			for (k0 = 0; k0 < K; k0++) {
				//v_fx = Dt[kk + k0];
				//v_fx = Dm[kk + k0] + m_fx0[k0] + m_fx2[k2];
				v_fx = Dm[kk + k0] + m_fx0_u[k0] + m_fx2_u[k2];
				TRUNCATE(delta_1, v_fx);
			}
		}
		//m_fx1_u[k1] = r_fx1 * (delta_1-delta_f) - m_fx1[k1];
		//m_fx1_u[k1] = r_fx1 * (delta_1-delta_f) + (r_fx1-1) * m_fx1[k1];
		m_fx_buf[k1] = r_fx1 * (delta_1-delta_f) + (r_fx1-1) * m_fx1_u[k1];
	}
	memcpy(m_fx1_b, m_fx_buf, K * sizeof(REALV));

	for (k2 = 0; k2 < K; k2++) {
		delta_2 = INFINITE_S;
		k2_K_2 = k2 * K_2;
		for (k1 = 0; k1 < K; k1++) {
			kk = k2_K_2 + k1 * K;
			for (k0 = 0; k0 < K; k0++) {
				//v_fx = Dt[kk + k0];
				//v_fx = Dm[kk + k0] + m_fx0[k0] + m_fx1[k1];
				v_fx = Dm[kk + k0] + m_fx0_u[k0] + m_fx1_u[k1];
				TRUNCATE(delta_2, v_fx);
			}
		}
		//m_fx2_u[k2] = r_fx2 * (delta_2-delta_f) - m_fx2[k2];
		//m_fx2_u[k2] = r_fx2 * (delta_2-delta_f) + (r_fx2-1) * m_fx2[k2];
		m_fx_buf[k2] = r_fx2 * (delta_2-delta_f) + (r_fx2-1) * m_fx2_u[k2];
	}
	memcpy(m_fx2_b, m_fx_buf, K * sizeof(REALV));
	//////////////////////////////////////////
	//////////////////////////////////////////

	//////////////////////////////////////////
	//////////////////////////////////////////
	memcpy(m_fx0_u, m_fx0_b, K * sizeof(REALV));
	memcpy(m_fx1_u, m_fx1_b, K * sizeof(REALV));
	memcpy(m_fx2_u, m_fx2_b, K * sizeof(REALV));

	cu_SubtractMin(m_fx0_u, m_fx0, K, delta_0);
	cu_SubtractMin(m_fx1_u, m_fx1, K, delta_1);
	cu_SubtractMin(m_fx2_u, m_fx2, K, delta_2);
	//////////////////////////////////////////
	//////////////////////////////////////////

	//////////////////////////////////////////
	//////////////////////////////////////////
	//lowerBound += delta_f + delta_0 + delta_1 + delta_2;
	atomicAdd(&dLowerBound, (int)(delta_f + delta_0 + delta_1 + delta_2));
	//////////////////////////////////////////
	//////////////////////////////////////////
}

extern "C"
void cu_TRW_S(int iter, REALV**** pdcv, REALV**** pOffset[3], REALV**** pRangeTerm[3], REALV**** pSO1[3][3], REALV**** pSO2[3][3],
	REALV**** pSO1F2Message[3][6], REALV**** pSO2F3Message[3][9], REALV**** pDualMessage[3], double* pLowerBound, double* pLowerBoundPrev, int iterPrev)
{
	int i, j, l, it;

#ifdef CU_USE_TIMER
#ifdef CU_USE_CUTIL
	unsigned int timer;
	cutCreateTimer(&timer);
	cutStartTimer(timer);
#else
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif
#endif

	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////
	if (iterPrev == 0) {
		if (mmode == 0) {
			cu_VolInit(ddcv, pdcv, 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, num_d);
		}
		for (i = 0; i < 3; i++) {
			cu_VolInit(hOffset[i], pOffset[i], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, 1);
			cu_VolInit(hRangeTerm[i], pRangeTerm[i], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, K);
			if (in_scv_w_O1F2 != -2) {
				if (mmode == 0) {
					for (j = 0; j < 3; j++) {
						cu_VolInit(hSO1[i][j], pSO1[i][j], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, L4_1);
					}
				}
				for (j = 0; j < 6; j++) {
					cu_VolInit(hSO1F2Message[i][j], pSO1F2Message[i][j], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, K);
				}
			}
			if (in_scv_w_O2F3 != -2) {
				if (mmode == 0) {
					for (j = 0; j < 3; j++) {
						cu_VolInit(hSO2[i][j], pSO2[i][j], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, L8_1);
					}
				}
				for (j = 0; j < 9; j++) {
					cu_VolInit(hSO2F3Message[i][j], pSO2F3Message[i][j], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, K);
				}
			}
			cu_VolInit(hDualMessage[i], pDualMessage[i], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, K);
		}
	}
	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////

#ifdef CU_USE_TIMER
#ifdef CU_USE_CUTIL
	cutStopTimer(timer);
	TRACE2("h->d time = %f\n", cutGetTimerValue(timer));
	cutDeleteTimer(timer);

	cutCreateTimer(&timer);
	cutStartTimer(timer);
#else
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	TRACE2("h->d time = %f\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif
#endif

	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////
	//cudaPrintfInit(blocksInX*blocksInY*blocksInZ*threadsInX*threadsInY*threadsInZ*256);

	for (it = 0; it < iter; it++) {
		iLowerBound = 0;
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dLowerBound), &iLowerBound, sizeof(unsigned long long int)));

		///////////////////////////////////////////////////////////////////////////////////////
		// forward update
		///////////////////////////////////////////////////////////////////////////////////////

		///////////////////////////////////////////////////////////////////////////////////////
		if ((in_scv_w_O1F2 != -2) && (in_scv_w_O2F3 == -2)) {
			for (l = 0; l < 3; l++) {
				cu_UpdateMessage_TRW_S_FW_O1F2<<<Dg, Db, Ns>>>(blocksInY, invBlocksInY, 0, 0, 0, mx, my, mz, hRangeTerm[l], ddSO1F2Message[l], hDualMessage[l]);
			}
		} else if ((in_scv_w_O1F2 == -2) && (in_scv_w_O2F3 != -2)) {
			for (l = 0; l < 3; l++) {
				cu_UpdateMessage_TRW_S_FW_O2F3<<<Dg, Db, Ns>>>(blocksInY, invBlocksInY, 0, 0, 0, mx, my, mz, hRangeTerm[l], ddSO2F3Message[l], hDualMessage[l]);
			}
		} else if ((in_scv_w_O1F2 != -2) && (in_scv_w_O2F3 != -2)) {
			for (l = 0; l < 3; l++) {
				cu_UpdateMessage_TRW_S_FW_O1F2_O2F3<<<Dg, Db, Ns>>>(blocksInY, invBlocksInY, 0, 0, 0, mx, my, mz, hRangeTerm[l], ddSO1F2Message[l], ddSO2F3Message[l], hDualMessage[l]);
			}
		}
		hipDeviceSynchronize();
		///////////////////////////////////////////////////////////////////////////////////////

		///////////////////////////////////////////////////////////////////////////////////////
		// backward update
		///////////////////////////////////////////////////////////////////////////////////////

		///////////////////////////////////////////////////////////////////////////////////////
		if (in_scv_w_O1F2 != -2) {
			for (l = 2; l >= 0; l--) {
				if (mmode == 1) {
					for (j = 0; j < 3; j++) {
						CUDA_SAFE_CALL(hipMalloc(&hSO1[l][j], msize * L4_1 * sizeof(REALV)));
						cu_VolInit(hSO1[l][j], pSO1[l][j], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, L4_1);
					}
					CUDA_SAFE_CALL(hipMemcpy(ddSO1[l], hSO1[l], 3 * sizeof(REALV*), hipMemcpyHostToDevice));
				}
				//
				cu_UpdateSpatialMessage_TRW_S_BW_O1F2<<<Dg, Db, Ns>>>(blocksInY, invBlocksInY, 4, 0, 0, 0, mx, my, mz, hOffset[l], ddSO1[l], ddSO1F2Message[l]);
				cu_UpdateSpatialMessage_TRW_S_BW_O1F2<<<Dg, Db, Ns>>>(blocksInY, invBlocksInY, 2, 0, 0, 0, mx, my, mz, hOffset[l], ddSO1[l], ddSO1F2Message[l]);
				cu_UpdateSpatialMessage_TRW_S_BW_O1F2<<<Dg, Db, Ns>>>(blocksInY, invBlocksInY, 0, 0, 0, 0, mx, my, mz, hOffset[l], ddSO1[l], ddSO1F2Message[l]);
				//
				if (mmode == 1) {
					for (j = 0; j < 3; j++) {
						CUDA_SAFE_CALL(hipFree(hSO1[l][j]));
					}
				}
			}
		}
		if (in_scv_w_O2F3 != -2) {
			for (l = 2; l >= 0; l--) {
				if (mmode == 1) {
					for (j = 0; j < 3; j++) {
						CUDA_SAFE_CALL(hipMalloc(&hSO2[l][j], msize * L8_1 * sizeof(REALV)));
						cu_VolInit(hSO2[l][j], pSO2[l][j], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, L8_1);
					}
					CUDA_SAFE_CALL(hipMemcpy(ddSO2[l], hSO2[l], 3 * sizeof(REALV*), hipMemcpyHostToDevice));
				}
				//
				cu_UpdateSpatialMessage_TRW_S_BW_O2F3<<<Dg, Db, Ns>>>(blocksInY, invBlocksInY, 7, 0, 0, 0, mx, my, mz, ddSO2[l], ddSO2F3Message[l]);
				cu_UpdateSpatialMessage_TRW_S_BW_O2F3<<<Dg, Db, Ns>>>(blocksInY, invBlocksInY, 4, 0, 0, 0, mx, my, mz, ddSO2[l], ddSO2F3Message[l]);
				cu_UpdateSpatialMessage_TRW_S_BW_O2F3<<<Dg, Db, Ns>>>(blocksInY, invBlocksInY, 1, 0, 0, 0, mx, my, mz, ddSO2[l], ddSO2F3Message[l]);
				//
				if (mmode == 1) {
					for (j = 0; j < 3; j++) {
						CUDA_SAFE_CALL(hipFree(hSO2[l][j]));
					}
				}
			}
		}
		hipDeviceSynchronize();
		///////////////////////////////////////////////////////////////////////////////////////

		///////////////////////////////////////////////////////////////////////////////////////
		//*
		if (mmode == 0) {
			cu_UpdateDualMessage_TRW_S_BW<<<Dg, Db, Ns>>>(blocksInY, invBlocksInY, 0, 0, 0, mx, my, mz, ddcv, dDualMessage);
		} else if (mmode == 1) {
			size_t msize_4 = msize / 4;
			int mz_4  =     mz / 4;
			int mz_42 = 2 * mz / 4;
			int mz_43 = 3 * mz / 4;
			//
			#if 0
			{
				size_t free_mem, total_mem, req_mem;
				CUDA_SAFE_CALL(hipMemGetInfo(&free_mem, &total_mem));
				req_mem = msize_4 * num_d * sizeof(REALV);
				if (req_mem > free_mem) {
					TRACE("req_mem = %u, free_mem = %u\n", req_mem, free_mem);
					return;
				}
			}
			#endif
			//
			CUDA_SAFE_CALL(hipMalloc(&ddcv, msize_4 * num_d * sizeof(REALV)));
			//
			cu_VolInit(ddcv, pdcv, 0, 0, 0    , mx, my, mz_4, mesh_x, mesh_y, mesh_z, num_d);
			cu_UpdateDualMessage_TRW_S_BW<<<Dg_4, Db, Ns>>>(blocksInY, invBlocksInY, 0, 0, 0    , mx, my, mz_4, ddcv, dDualMessage);
			//
			cu_VolInit(ddcv, pdcv, 0, 0, mz_4 , mx, my, mz_4, mesh_x, mesh_y, mesh_z, num_d);
			cu_UpdateDualMessage_TRW_S_BW<<<Dg_4, Db, Ns>>>(blocksInY, invBlocksInY, 0, 0, mz_4 , mx, my, mz_4, ddcv, dDualMessage);
			//
			cu_VolInit(ddcv, pdcv, 0, 0, mz_42, mx, my, mz_4, mesh_x, mesh_y, mesh_z, num_d);
			cu_UpdateDualMessage_TRW_S_BW<<<Dg_4, Db, Ns>>>(blocksInY, invBlocksInY, 0, 0, mz_42, mx, my, mz_4, ddcv, dDualMessage);
			//
			cu_VolInit(ddcv, pdcv, 0, 0, mz_43, mx, my, mz_4, mesh_x, mesh_y, mesh_z, num_d);
			cu_UpdateDualMessage_TRW_S_BW<<<Dg_4, Db, Ns>>>(blocksInY, invBlocksInY, 0, 0, mz_43, mx, my, mz_4, ddcv, dDualMessage);
			//
			hipFree(ddcv);
		}
		//*/
		hipDeviceSynchronize();
		///////////////////////////////////////////////////////////////////////////////////////

		///////////////////////////////////////////////////////////////////////////////////////
		for (l = 2; l >= 0; l--) {
			cu_UpdateMessage_TRW_S_BW<<<Dg, Db, Ns>>>(blocksInY, invBlocksInY, 0, 0, 0, mx, my, mz, hRangeTerm[l], ddSO1F2Message[l], ddSO2F3Message[l], hDualMessage[l]);
			//
			//FILE* fp = fopen("culog.txt", "w");
			//cudaPrintfDisplay(fp, false);
			//fflush(fp);
			//fclose(fp);
		}
		hipDeviceSynchronize();
		///////////////////////////////////////////////////////////////////////////////////////

		CUDA_SAFE_CALL(hipMemcpyFromSymbol(&iLowerBound, HIP_SYMBOL(dLowerBound), sizeof(unsigned long long int)));
		*pLowerBound = (double)iLowerBound;

		if (it < iter-1) {
			TRACE2("iter %03d, lb = %f, lb_delta = %f\n", it+iterPrev, *pLowerBound, *pLowerBound-*pLowerBoundPrev);
			//
			*pLowerBoundPrev = *pLowerBound;
		}
	}

	//cudaPrintfEnd();
	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////

#ifdef CU_USE_TIMER
#ifdef CU_USE_CUTIL
	cutStopTimer(timer);
	TRACE2("avg time = %f\n", cutGetTimerValue(timer) / iter);
	cutDeleteTimer(timer);

	cutCreateTimer(&timer);
	cutStartTimer(timer);
#else
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	TRACE2("avg time = %f\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif
#endif

	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////
	for (i = 0; i < 3; i++) {
		if (in_scv_w_O1F2 != -2) {
			for (j = 0; j < 6; j++) {
				cu_VolCopy(hSO1F2Message[i][j], pSO1F2Message[i][j], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, K);
			}
		}
		if (in_scv_w_O2F3 != -2) {
			for (j = 0; j < 9; j++) {
				cu_VolCopy(hSO2F3Message[i][j], pSO2F3Message[i][j], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, K);
			}
		}
		cu_VolCopy(hDualMessage[i], pDualMessage[i], 0, 0, 0, mx, my, mz, mesh_x, mesh_y, mesh_z, K);
	}
	///////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////

#ifdef CU_USE_TIMER
#ifdef CU_USE_CUTIL
	cutStopTimer(timer);
	TRACE2("d->h time = %f\n", cutGetTimerValue(timer));
	cutDeleteTimer(timer);
#else
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	TRACE2("d->h time = %f\n", elapsedTime);
   	hipEventDestroy(start);
	hipEventDestroy(stop); 
#endif
#endif
}
///////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////
